#include "hip/hip_runtime.h"
#include <stdio.h>
#include "mpi.h"
#include "nvshmem.h"
#include "nvshmemx.h"

#define SIZE 1

#define CUDA_CHECK(stmt)                                  \
do {                                                      \
    hipError_t result = (stmt);                          \
    if (hipSuccess != result) {                          \
        fprintf(stderr, "[%s:%d] CUDA failed with %s \n", \
         __FILE__, __LINE__, hipGetErrorString(result)); \
        exit(-1);                                         \
    }                                                     \
} while (0)

__global__ void simple_shift(int *destination) {
    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    int peer = (mype + 1) % npes;
    

    for (int i = 0; i < SIZE; i++) {
        nvshmem_int_p(destination + i * sizeof(int), mype, peer);
    }
}

int main (int argc, char *argv[]) {
    int mype_node, msg;
    hipStream_t stream;
    int rank, nranks;
    MPI_Comm mpi_comm = MPI_COMM_WORLD;
    nvshmemx_init_attr_t attr;

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &nranks);

    attr.mpi_comm = &mpi_comm;
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);

    CUDA_CHECK(hipSetDevice(mype_node));
    CUDA_CHECK(hipStreamCreate(&stream));
    int *destination = (int *) nvshmem_malloc (SIZE * sizeof(int));

    simple_shift<<<1, 1, 0, stream>>>(destination);
    nvshmemx_barrier_all_on_stream(stream);
    CUDA_CHECK(hipMemcpyAsync(&msg, destination, sizeof(int),
                hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));
    printf("%d: received message %d\n", nvshmem_my_pe(), msg);

    nvshmem_free(destination);
    nvshmem_finalize();
    MPI_Finalize();
    return 0;
}