#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include<unistd.h>

#include "cuda_wrapper.hpp"

using namespace std;


#define gpuErrchk(ans) { gpuAssert(ans); }
inline void gpuAssert(hipError_t code)
{
  if (code != hipSuccess) {
  fprintf(stderr,"GPUassert: %s\n", hipGetErrorString(code));
  }
}

__host__ __device__ int modify_cell(int a) {
  return a + 2;
}

__global__ void simple_kernel(int *src, int *dst1){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  dst1[idx] = modify_cell(src[idx]);
}

int main() {
  int gpuid[] = {0, 1};

  int canAccessPeer;

  hipError_t err = hipDeviceCanAccessPeer(&canAccessPeer, 0, 1);
  if (err != hipSuccess) {
  }

  hipSetDevice(gpuid[0]);
  gpuErrchk(hipDeviceEnablePeerAccess(1, 0));
  hipSetDevice(gpuid[1]);
  gpuErrchk(hipDeviceEnablePeerAccess(0, 0));

  const size_t size = 32;
  const size_t buf_size = size * sizeof(int);

  int *g0 = NULL;
  hipSetDevice(gpuid[0]);
  gpuErrchk(cudaMallocWRAP(&g0, buf_size, "g0", 4));

  //gpuErrchk(hipMalloc(&g0, buf_size));

  int *g1 = NULL;

  hipSetDevice(gpuid[1]);
  gpuErrchk(cudaMallocWRAP(&g1, buf_size, "g1", 4));
  //gpuErrchk(hipMalloc(&g1, buf_size));
  hipSetDevice(gpuid[0]);

  int *h0 = NULL;
  gpuErrchk(hipHostMalloc(&h0, buf_size));

  hipSetDevice(gpuid[1]);
  int *h1 = NULL;
  gpuErrchk(hipHostMalloc(&h1, buf_size));

  hipSetDevice(gpuid[0]);
  gpuErrchk(hipMemcpy(g0, h0, buf_size, hipMemcpyHostToDevice));

  simple_kernel<<<1, size>>>(g0, g1);
  //hipDeviceSynchronize();

  gpuErrchk(hipMemcpy(h1, g1, buf_size, hipMemcpyDeviceToHost));
//#if 0
  hipSetDevice(gpuid[1]);
  gpuErrchk(hipMemcpy(g1, h1, buf_size, hipMemcpyHostToDevice));

  simple_kernel<<<1, size>>>(g1, g0);
  //hipDeviceSynchronize();

  gpuErrchk(hipMemcpy(h0, g0, buf_size, hipMemcpyDeviceToHost));
//#endif
  hipFree(h0);
  hipFree(h1);
  hipFree(g0);
  hipFree(g1);

  return 0;
}
