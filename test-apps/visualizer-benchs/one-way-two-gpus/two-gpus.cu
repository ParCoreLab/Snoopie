#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include<unistd.h>

#include "cuda_wrapper.hpp"

using namespace std;


#define gpuErrchk(ans) { gpuAssert(ans); }
inline void gpuAssert(hipError_t code)
{
  if (code != hipSuccess) {
  fprintf(stderr,"GPUassert: %s\n", hipGetErrorString(code));
  }
}

__host__ __device__ int modify_cell(int a) {
  return a + 2;
}

__global__ void simple_kernel(int *src, int *dst1){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  dst1[idx] = modify_cell(src[idx]);
}

int main() {
  int gpuid[] = {0, 1};

  int canAccessPeer;
  hipError_t err = hipDeviceCanAccessPeer(&canAccessPeer, 0, 1);
  if (err != hipSuccess) {
  }
  gpuErrchk(hipDeviceEnablePeerAccess(1, 0));
  gpuErrchk(hipDeviceEnablePeerAccess(2, 0));

  const size_t size = 32;
  const size_t buf_size = size * sizeof(int);

  int *g0 = NULL;
  hipSetDevice(gpuid[0]);
  gpuErrchk(cudaMallocWRAP(&g0, buf_size, "g0", 4));

  int *g1 = NULL;

  hipSetDevice(gpuid[1]);
  gpuErrchk(cudaMallocWRAP(&g1, buf_size, "g1", 4));
  hipSetDevice(gpuid[0]);

  int *h0 = NULL;
  gpuErrchk(cudaMallocHostWRAP(&h0, buf_size, "h0", 4));

  hipSetDevice(gpuid[1]);
  int *h1 = NULL;
  gpuErrchk(cudaMallocHostWRAP(&h1, buf_size, "h1", 4));

  gpuErrchk(hipMemcpy(g0, h0, buf_size, hipMemcpyHostToDevice));

  hipSetDevice(gpuid[0]);
  simple_kernel<<<1, size>>>(g0, g1);

  gpuErrchk(hipMemcpy(h1, g1, buf_size, hipMemcpyDeviceToHost));
  
  hipFree(h0);
  hipFree(h1);
  hipFree(g0);
  hipFree(g1);

  return 0;
}
