#include <stdio.h>
#include "cuda_wrapper.hpp"

//#include <experimental/source_location>
//hipError_t hipMalloc ( void** devPtr, size_t size, const std::experimental::source_location& location = std::experimental::source_location::current());
//#define cudaMallocWRAP(a, b, var_name) cudaMallocWrap((void **)a, b, (char *) var_name, __FILE__, __func__, __LINE__)

int main(){

  int *d_a[4], *d_b[6];
  for(int i = 0; i < 4; i++)
  	cudaMallocWRAP(&d_a[i], 2 * sizeof(*d_a[0]), "d_a");
  for(int i = 0; i < 6; i++)
        cudaMallocWRAP(&d_b[i], 4 * sizeof(*d_b[0]), "d_b");
  for(int i = 0; i < 4; i++)
  	hipFree(&d_a[i]);
  for(int i = 0; i < 6; i++)
        hipFree(&d_b[i]);
  return 0;
}
