
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include<unistd.h>

using namespace std;


#define gpuErrchk(ans) { gpuAssert(ans); }
inline void gpuAssert(hipError_t code)
{
  if (code != hipSuccess) {
  fprintf(stderr,"GPUassert: %s\n", hipGetErrorString(code));
  }
}

__host__ __device__ int modify_cell(int a) {
  return a + 2;
}

__global__ void simple_kernel(int *src, int *dst1, int *dst2){
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx % 2 == 0) {
    dst1[idx] = modify_cell(src[idx]);
  }
  else {
    dst2[idx] = modify_cell(src[idx]);
  }
}

int main() {
  int gpuid[] = {0, 1, 2};

  int canAccessPeer;
  hipError_t err = hipDeviceCanAccessPeer(&canAccessPeer, 0, 1);
  if (err != hipSuccess) {
  }
  gpuErrchk(hipDeviceEnablePeerAccess(1, 0));
  gpuErrchk(hipDeviceEnablePeerAccess(2, 0));
  gpuErrchk(hipDeviceEnablePeerAccess(3, 0));

  const size_t size = 32;
  const size_t buf_size = size * sizeof(int);

  int *g0 = NULL;
  hipSetDevice(gpuid[0]);
  gpuErrchk(hipMalloc(&g0, buf_size));

  int *g1 = NULL;

  hipSetDevice(gpuid[1]);
  gpuErrchk(hipMalloc(&g1, buf_size));
  hipSetDevice(gpuid[0]);

  int *g2 = NULL;

  hipSetDevice(gpuid[2]);
  gpuErrchk(hipMalloc(&g2, buf_size));
  hipSetDevice(gpuid[0]);

  int *h0 = NULL;
  gpuErrchk(hipHostMalloc(&h0, buf_size, hipHostMallocDefault));

  int *h1 = NULL;
  gpuErrchk(hipHostMalloc(&h1, buf_size, hipHostMallocDefault));

  int *h2 = NULL;
  gpuErrchk(hipHostMalloc(&h2, buf_size, hipHostMallocDefault));

  gpuErrchk(hipMemcpy(g0, h0, buf_size, hipMemcpyHostToDevice));

  hipSetDevice(gpuid[0]);
  simple_kernel<<<1, size>>>(g0, g1, g2);
  gpuErrchk(hipMemcpy(h1, g1, buf_size, hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(h2, g2, buf_size, hipMemcpyDeviceToHost));

  for (int i = 0; i < size; i++) {
    //printf("\rchecking correctness against CPU: %.2f", ((float) i / (float) size) * 100);
	  if (i % 2 == 0 && h1[i] == modify_cell(h0[i])) {
		  continue;
    } else if (i % 2 == 1 && h2[i] == modify_cell(h0[i])) {
		  continue;
    }

    cout << "FAILED: modify_cell((H0: " << i << ")) " << modify_cell(h0[i]) << "  != (H1: " << i << ") " << h1[i] << endl;
    return 1;
  }

  //printf("\ntransfer finished successfully\n");

  
  hipFree(h0);
  hipFree(h1);
  hipFree(h2);
  hipFree(g0);
  hipFree(g1);
  hipFree(g2);

  return 0;
}
