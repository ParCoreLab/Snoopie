
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <unistd.h>
#include <getopt.h>

using namespace std;


#define gpuErrchk(ans) { gpuAssert(ans); }
inline void gpuAssert(hipError_t code)
{
  if (code != hipSuccess) {
  fprintf(stderr,"GPUassert: %s\n", hipGetErrorString(code));
  }
}

__global__ void set_gpu_vals(int size, int *arr, int val){
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;

  arr[idx] = val;
}

struct hipa_args {
  int size = 32;
  int verbose = 0;
  int check = 0;
  int async = 0;
};

typedef struct hipa_args hipa_args;


void getargs(hipa_args *args, int argc, char* argv[]) {
  int c;

  while ((c = getopt(argc, argv, "n:avc")) != -1) {
    switch (c) {
      case 'n':
        args->size = atoi(optarg);
        if (args->size <= 0) {
          fprintf(stderr, "Error: argument for -n cannot be 0 or less\n");
        }
        break;
      case 'v':
        args->verbose = 1;
        break;
      case 'a':
        args->async = 1;
        break;
      case 'c':
        args->check = 1;
        break;
      case '?':
        if (optopt == 'n') {
          fprintf(stderr, "Error: no argument provided for -n flag\n");
        } else {
          fprintf(stderr, "Error: unknown option '%c'\n", optopt);
        }
        exit(1);
      default:
        abort();
    }
  }
}

hipa_args *default_args() {
  hipa_args *args = (hipa_args*) malloc(sizeof(hipa_args));

  args->size = 32;
  args->verbose = 0;
  args->check = 0;
  args->async = 0;
  
  return args;
}

int main(int argc, char* argv[]) {

  hipa_args *args = default_args();
  getargs(args, argc, argv);

  int gpuid[] = {0, 1};

  hipSetDevice(gpuid[0]);
  gpuErrchk(hipDeviceEnablePeerAccess(1, 0));
  hipSetDevice(gpuid[1]);
  gpuErrchk(hipDeviceEnablePeerAccess(0, 0));
  hipSetDevice(gpuid[0]);

  const size_t buf_size = args->size * sizeof(int);

  int *g0 = NULL;
  hipSetDevice(gpuid[0]);
  gpuErrchk(hipMalloc(&g0, buf_size));
  hipSetDevice(gpuid[0]);

  int *g1 = NULL;
  hipSetDevice(gpuid[1]);
  gpuErrchk(hipMalloc(&g1, buf_size));
  hipSetDevice(gpuid[0]);

  int *h0 = NULL;
  gpuErrchk(hipHostMalloc(&h0, buf_size, hipHostMallocDefault));

  int *h1 = NULL;
  gpuErrchk(hipHostMalloc(&h1, buf_size, hipHostMallocDefault));

  set_gpu_vals<<<std::ceil(args->size / 1024.0), max(args->size > 1024 ? 1024 :args->size % 1025, 1)>>>(args->size, g0, 10);
  gpuErrchk(hipDeviceSynchronize());

  if (args->async) {
    gpuErrchk(hipMemcpyAsync(g1, g0, buf_size, hipMemcpyDeviceToDevice));
  } else {
    gpuErrchk(hipMemcpy(g1, g0, buf_size, hipMemcpyDeviceToDevice));
  }

  if (args->check) {
    gpuErrchk(hipMemcpy(h0, g0, buf_size, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h1, g1, buf_size, hipMemcpyDeviceToHost));

    for (int i = 0; i < args->size; i++) {
      if (args->verbose) {
        printf("\rchecking correctness against CPU: %.2f", ((float) (i + 1) / (float) args->size) * 100);

        if (i == args->size - 1) {
          printf("\n");
        }
      }

      if (h1[i] == h0[i]) {
        continue;
      }

      cout << "FAILED: (H0: " << i << ") " << h0[i] << "  != (H1: " << i << ") " << h1[i] << endl;
      return 1;
    }
  }

  
  hipFree(h0);
  hipFree(h1);
  hipFree(g0);
  hipFree(g1);

  free(args);

  return 0;
}
