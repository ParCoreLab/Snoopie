#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include<unistd.h>

#include "cuda_wrapper.hpp"

using namespace std;


#define gpuErrchk(ans) { gpuAssert(ans); }
inline void gpuAssert(hipError_t code)
{
  if (code != hipSuccess) {
  fprintf(stderr,"GPUassert: %s\n", hipGetErrorString(code));
  }
}

__host__ __device__ int modify_cell(int a) {
  return a + 2;
}

__global__ void simple_kernel(int *src, int *dst1){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  dst1[idx] = modify_cell(src[idx]);
}

int main() {
  int gpuid[] = {0, 1, 2};

  int canAccessPeer;
  hipError_t err = hipDeviceCanAccessPeer(&canAccessPeer, 0, 1);
  if (err != hipSuccess) {
  }
 
  hipSetDevice(gpuid[0]);
  gpuErrchk(hipDeviceEnablePeerAccess(1, 0));
  gpuErrchk(hipDeviceEnablePeerAccess(2, 0));
  gpuErrchk(hipDeviceEnablePeerAccess(3, 0));
   

  const size_t size = 32;
  const size_t buf_size = size * sizeof(int);

  int *g0 = NULL;
  hipSetDevice(gpuid[0]);
  gpuErrchk(cudaMallocWRAP(&g0, buf_size, "g0", 4));

  int *g1 = NULL;

  hipSetDevice(gpuid[1]);
  gpuErrchk(cudaMallocWRAP(&g1, buf_size, "g1", 4));

  int *g2 = NULL;

  hipSetDevice(gpuid[2]);
  gpuErrchk(cudaMallocWRAP(&g2, buf_size, "g2", 4));

  int *g3 = NULL;

  hipSetDevice(gpuid[3]);
  gpuErrchk(cudaMallocWRAP(&g3, buf_size, "g3", 4));


  hipSetDevice(gpuid[0]);

  int *h0 = NULL;
  gpuErrchk(cudaMallocHostWRAP(&h0, buf_size, "h0", 4));

  hipSetDevice(gpuid[1]);
  int *h1 = NULL;
  gpuErrchk(cudaMallocHostWRAP(&h1, buf_size, "h1", 4));

  hipSetDevice(gpuid[2]);
  int *h2 = NULL;
  gpuErrchk(cudaMallocHostWRAP(&h2, buf_size, "h2", 4));

  hipSetDevice(gpuid[3]);
  int *h3 = NULL;
  gpuErrchk(cudaMallocHostWRAP(&h3, buf_size, "h3", 4));

  hipSetDevice(gpuid[0]);
  gpuErrchk(hipMemcpy(g0, h0, buf_size, hipMemcpyHostToDevice));

  simple_kernel<<<1, size>>>(g0, g1);
  simple_kernel<<<1, size>>>(g0, g2);
  simple_kernel<<<1, size>>>(g0, g3);
  
  gpuErrchk(hipMemcpy(h1, g1, buf_size, hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(h2, g2, buf_size, hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(h3, g3, buf_size, hipMemcpyDeviceToHost));

  hipFree(h0);
  hipFree(h1);
  hipFree(h2);
  hipFree(h3);
  hipFree(g0);
  hipFree(g1);
  hipFree(g2);
  hipFree(g3);

  return 0;
}
