#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include<unistd.h>

#include "cuda_wrapper.hpp"

using namespace std;


#define gpuErrchk(ans) { gpuAssert(ans); }
inline void gpuAssert(hipError_t code)
{
  if (code != hipSuccess) {
  fprintf(stderr,"GPUassert: %s\n", hipGetErrorString(code));
  }
}

__host__ __device__ int modify_cell(int a) {
  return a + 2;
}

__global__ void simple_kernel(int *dst, int *src1, int *src2, int *src3){
  //int idx = blockIdx.x * blockDim.x + threadIdx.x;
  dst[50] = modify_cell(src1[50]);
  dst[50] += modify_cell(src2[50]);
  dst[50] += modify_cell(src3[50]);
}

int main() {
  int gpuid[] = {0, 1, 2, 3};

  int canAccessPeer;
  hipError_t err = hipDeviceCanAccessPeer(&canAccessPeer, 0, 1);
  if (err != hipSuccess) {
  }
 
  hipSetDevice(gpuid[0]);
  gpuErrchk(hipDeviceEnablePeerAccess(1, 0));
  gpuErrchk(hipDeviceEnablePeerAccess(2, 0));
  gpuErrchk(hipDeviceEnablePeerAccess(3, 0));
   

  const size_t size = 100;
  const size_t buf_size = size * sizeof(int);

  int *g0 = NULL;
  int *g1 = NULL;
  int *g2 = NULL;
  int *g3 = NULL;
  int *h0 = NULL;
  int *h1 = NULL;
  int *h2 = NULL;
  int *h3 = NULL;

  //hipSetDevice(gpuid[0]);
  gpuErrchk(cudaMallocWRAP(&g0, buf_size, "g0", 4));
  gpuErrchk(cudaMallocHostWRAP(&h0, buf_size, "h0", 4));

  hipSetDevice(gpuid[1]);
  gpuErrchk(cudaMallocWRAP(&g1, buf_size, "g1", 4));
  gpuErrchk(cudaMallocHostWRAP(&h1, buf_size, "h1", 4));
  gpuErrchk(hipMemcpy(g1, h1, buf_size, hipMemcpyHostToDevice));

  hipSetDevice(gpuid[2]);
  gpuErrchk(cudaMallocWRAP(&g2, buf_size, "g2", 4));
  gpuErrchk(cudaMallocHostWRAP(&h2, buf_size, "h2", 4));
  gpuErrchk(hipMemcpy(g2, h2, buf_size, hipMemcpyHostToDevice));

  hipSetDevice(gpuid[3]);
  gpuErrchk(cudaMallocWRAP(&g3, buf_size, "g3", 4));
  gpuErrchk(cudaMallocHostWRAP(&h3, buf_size, "h3", 4));
  gpuErrchk(hipMemcpy(g3, h3, buf_size, hipMemcpyHostToDevice));

  hipSetDevice(gpuid[0]);

  simple_kernel<<<1, size>>>(g0, g1, g2, g3);
  
  gpuErrchk(hipMemcpy(h0, g0, buf_size, hipMemcpyDeviceToHost));

  hipFree(h0);
  hipFree(h1);
  hipFree(h2);
  hipFree(h3);
  hipFree(g0);
  hipFree(g1);
  hipFree(g2);
  hipFree(g3);

  return 0;
}
