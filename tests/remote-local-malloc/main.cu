#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include <stdio.h>
#include <unistd.h>
#include <cuda/std/chrono>

using namespace std;

// __device__ volatile int *shared_ptr = NULL;

#define gpuErrchk(ans) { gpuAssert(ans); }

__host__ __device__ inline void gpuAssert(hipError_t code)
{
  if (code != hipSuccess) {
    printf("GPUassert: %d\n", code);
  }
}

__global__ void alloc_spin(int **shared_ptr){
  gpuErrchk(hipMalloc(shared_ptr, sizeof(int)));
  printf("*shared ptr is now: %p\n", *shared_ptr);
  **shared_ptr = 10;
}

__global__ void read_spin(int **shared_ptr){
  printf("Attempting to dereference shared ptr -> ");
  printf("*shared_ptr: %p\n", *shared_ptr);
  while(shared_ptr == NULL) { }
  printf("i: %d\n", **shared_ptr);
}


int main() {


  hipStream_t stream1;
  hipStreamCreate(&stream1);


  hipSetDevice(0);
  int **shared_ptr;
  hipMalloc(&shared_ptr, sizeof(*shared_ptr));
  // set_null<<<1, 1, 1, stream1>>>(shared_ptr);
  // hipDeviceSynchronize();

  alloc_spin<<<1, 1, 1, stream1>>>(shared_ptr);
  hipStream_t stream2;
  hipStreamCreate(&stream2);

  read_spin<<<1, 1, 1, stream2>>>(shared_ptr);

  hipStreamSynchronize(stream1);
  hipStreamSynchronize(stream2);

  gpuErrchk(hipGetLastError());

  return 0;
}