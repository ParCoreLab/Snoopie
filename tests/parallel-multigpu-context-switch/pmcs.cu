#include <iostream>
#include <omp.h>
#include <hip/hip_runtime.h>

#define CUDA_RT_CALL(call)                                                                  \
    {                                                                                       \
        hipError_t cudaStatus = call;                                                      \
        if (hipSuccess != cudaStatus)                                                      \
            fprintf(stderr,                                                                 \
                    "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "              \
                    "with "                                                                 \
                    "%s (%d).\n",                                                           \
                    #call, __LINE__, __FILE__, hipGetErrorString(cudaStatus), cudaStatus); \
    }                                                                                       \
    
int main() {
    std::cout << "Started multi context in parallel" << std::endl;

    int num_devices = 0;
    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

#pragma omp parallel num_threads(num_devices)
    {
        int dev_id = omp_get_thread_num();
        std::cout << "Switching to device: " << dev_id << std::endl;
#pragma omp barrier
        CUDA_RT_CALL(hipSetDevice(dev_id));
        // CUDA_RT_CALL(cudaFree(0));
        hipCtx_t ctx;
        hipDevicePrimaryCtxRetain(&ctx, dev_id);
#pragma omp barrier
        std::cout << "Switched to device: "  << dev_id << std::endl;
    }

    std::cout << "Ended multi context in parallel" << std::endl;

    return 0;
}
