#include "hip/hip_runtime.h"
#include "nvshmem.h"
#include "nvshmemx.h"

extern __host__ __device__ int modify_cell(int a);

__global__ void simple_kernel1(int size, int *data){
    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    int peer = ((mype - 1) >= 0) ? (mype - 1)  : (npes - 1);

    nvshmem_int_p(data, mype, peer); 
}

