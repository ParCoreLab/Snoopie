#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <dlfcn.h>
#include <iostream>
#include <map>
#include <sstream>
#include <stdint.h>
#include <stdio.h>
#include <string>
#include <unistd.h>
#include <unordered_map>
#include <unordered_set>

#include <adm_common.h>
#include <adm_config.h>
#include <adm_database.h>
#include <adm_memory.h>
#include <adm_splay.h>
#include <cpptrace/cpptrace.hpp>

#include "Python.h"
//#include "ndarrayobject.h"
#include <numpy/ndarrayobject.h>
#include <pybind11/pybind11.h>
#include <pybind11/embed.h>
#include <pybind11/pytypes.h>

#include <iostream>

/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

/* for channel */
#include "utils/channel.hpp"

/* contains definition of the mem_access_t structure */
#include "common.h"
#include "util.h"

#include "mpi.h"
#include "nvshmem.h"
#include "nvshmemx.h"

#include <sys/syscall.h>
#define gettid() syscall(SYS_gettid)

#define HEX(x)                                                                 \
	"0x" << std::setfill('0') << std::setw(16) << std::hex << (uint64_t)x        \
<< std::dec

#define CHANNEL_SIZE (1l << 30)

#define JSON 0
#define EQUAL_STRS 0

#define FILE_NAME_SIZE 256
#define PATH_NAME_SIZE 5000

#include "adm.h"

namespace py = pybind11;
using namespace cpptrace;
using namespace adamant;
using namespace std;

#define CHILD 1
#define SIBLING 2

int object_counter = 0;
int context_counter = 0;
int latest_context = 0;
static long mem_alloc_count = 0;

static bool nvshmem_malloc_handled = false;
static bool object_attribution = false;
pool_t<adm_splay_tree_t, ADM_DB_OBJ_BLOCKSIZE> *nodes = nullptr;
pool_t<adm_range_t, ADM_DB_OBJ_BLOCKSIZE> *ranges = nullptr;
static int global_index = 0;

static allocation_site_t *root = NULL;

static allocation_line_hash_table_t *allocation_line_table;

static execution_site_t *exec_root = NULL;

static execution_site_hash_table_t *execution_site_table;

std::vector<adm_range_t *> range_nodes;

std::vector<adm_object_t *> object_nodes;

std::vector<execution_context_t *> context_nodes;

Logger logger("snoopie_log_" + std::to_string(getpid()) + ".zst");

std::map<std::string, std::tuple<std::string, std::vector<int>, std::vector<int>>> line_tracking;

void initialize_object_table(int size);

void initialize_line_table(int size);

bool line_exists(int index);

std::string get_line_file_name(int index);

std::string get_line_dir_name(int index);

std::string get_line_sass(int index);

uint32_t get_line_line_num(int index);

short get_line_estimated_status(int index);

std::string get_object_var_name(uint64_t pc);

std::string get_object_file_name(uint64_t pc);

std::string get_object_func_name(uint64_t pc);

uint32_t get_object_line_num(uint64_t pc);

int get_object_device_id(uint64_t pc);

void set_object_device_id(uint64_t pc, int dev_id);

uint32_t get_object_data_type_size(uint64_t pc);

void set_object_data_type_size(uint64_t pc, const uint32_t type_size);

bool object_exists(uint64_t pc);
/* lock */
pthread_mutex_t mutex1;
//pthread_mutex_t mutex_pytorch;

/* map to store context state */
std::unordered_map<hipCtx_t, CTXstate *> ctx_state_map;

/* skip flag used to avoid re-entry on the nvbit_callback when issuing
 * flush_channel kernel call */
bool skip_callback_flag = false;

/* global control variables for this tool */
uint32_t instr_begin_interval = 0;
uint32_t instr_end_interval = UINT32_MAX;
std::string kernel_name;
std::string profiled_nccl_file = "";
int on_dev_filtering = 1;
int time_log = 0;
int verbose = 0;
std::string nvshmem_version = "2.8";
int nvshmem_ngpus = 10;
int silent = 0;
int code_attribution = 0;
int code_context = 0;
int data_object_attribution = 0;
int sample_size;

/* opcode to id map and reverse map  */
std::map<std::string, int> opcode_to_id_map;
std::map<int, std::string> id_to_opcode_map;
std::vector<MemoryAllocation> mem_allocs;

allocation_site_t *search_at_level(allocation_site_t *allocation_site, uint64_t pc) {
	if (allocation_site == NULL || allocation_site->get_pc() == pc)
		return allocation_site;

	return search_at_level(allocation_site->get_next_sibling(), pc);
}

execution_site_t *search_site_at_level(execution_site_t *execution_site, uint64_t exec_site_id) {
	if (execution_site == NULL || execution_site->get_exec_site_id() == exec_site_id)
		return execution_site;
	return search_site_at_level(execution_site->get_next_sibling(), exec_site_id);
}

std::map<unsigned long long, py::cpp_function> cur_tensorto_func;
int tensorto_func_count = 0;

void update_allocation_site_tree(py::object& summary, allocation_site_t **allocation_site, allocation_site_t **parent)
{
	std::vector<py::handle> stack_vec;
	for (py::handle frame : summary) {
		stack_vec.push_back(frame);
                if (root == NULL) {
			std::string filename = frame.attr("filename").attr("__str__")().cast<std::string>();
			uint64_t key_num = std::hash<std::string>()(filename);
			root = new allocation_site_t(key_num);
			*allocation_site = root;
		}
	}
	*parent = root;
        *allocation_site = root->get_first_child();
	
	for (auto itr = stack_vec.rbegin(); itr != stack_vec.rend(); ++itr) {
		std::cerr << itr->attr("filename").attr("__str__")().cast<std::string>() << " " << itr->attr("lineno").attr("__int__")().cast<int>() << " " << itr->attr("name").attr("__str__")().cast<std::string>() << std::endl;

		std::string filename = itr->attr("filename").attr("__str__")().cast<std::string>();
		int lineno = itr->attr("lineno").attr("__int__")().cast<int>();
		std::string key_str = filename + ":" + std::to_string(lineno);
		uint64_t key_num = std::hash<std::string>()(key_str);
		std::string func_name = itr->attr("name").attr("__str__")().cast<std::string>();

		allocation_line_t *line = allocation_line_table->find(key_num);
		if (line == NULL) {
			allocation_line_table->insert(new allocation_line_t(
				key_num, func_name, filename, lineno));
                }
		allocation_site_t *temp = *allocation_site;
		*allocation_site = search_at_level(*allocation_site, key_num);

		if (*allocation_site == NULL) {
			if (temp != NULL) {
				while (temp->get_next_sibling() != NULL)
					temp = temp->get_next_sibling();
				temp->set_next_sibling(new allocation_site_t(key_num));

				*allocation_site = temp->get_next_sibling();
				(*allocation_site)->set_parent(temp->get_parent());
			} else {
				(*parent)->set_first_child(new allocation_site_t(key_num));
				*allocation_site = (*parent)->get_first_child();
				(*allocation_site)->set_parent(*parent);
			}
		}
		*parent = *allocation_site;
		*allocation_site = (*allocation_site)->get_first_child();
	}
}

void update_exec_site_tree(py::object& summary, execution_site_t **execution_site, execution_site_t **parent)
{
	std::vector<py::handle> stack_vec;
	//execution_site_t *execution_site = NULL;
	//execution_site_t *parent = NULL; 
	for (py::handle frame : summary) {
		stack_vec.push_back(frame);
		if(exec_root == NULL) {
			std::string filename = frame.attr("filename").attr("__str__")().cast<std::string>();
			uint64_t key_num = std::hash<std::string>()(filename);
			exec_root = new execution_site_t(key_num);
			*execution_site = exec_root;
		}
	}
	*parent = exec_root;
	*execution_site = exec_root->get_first_child();
	
	for (auto itr = stack_vec.rbegin(); itr != stack_vec.rend(); ++itr) {
		std::string filename = itr->attr("filename").attr("__str__")().cast<std::string>();
		int lineno = itr->attr("lineno").attr("__int__")().cast<int>();
		std::string key_str = filename + ":" + std::to_string(lineno);
		uint64_t key_num = std::hash<std::string>()(key_str);
		execution_site_t *line = execution_site_table->find(key_num);
		if (line == NULL) {
			execution_site_table->insert(new execution_site_t(
				key_num, /*func_name,*/ filename, lineno));
		}
		execution_site_t *temp = *execution_site;
		*execution_site = search_site_at_level(*execution_site, key_num);

		if (*execution_site == NULL) {
			if (temp != NULL) {
				while (temp->get_next_sibling() != NULL)
					temp = temp->get_next_sibling();
				temp->set_next_sibling(new execution_site_t(key_num));
				*execution_site = temp->get_next_sibling();
				(*execution_site)->set_parent(temp->get_parent());
			} else {
				(*parent)->set_first_child(new execution_site_t(key_num));
				*execution_site = (*parent)->get_first_child();
				(*execution_site)->set_parent(*parent);
			}
		}
		*parent = *execution_site;
		*execution_site = (*execution_site)->get_first_child();
	}
}

//#if 0
inline void update_exec_site_tree_cpp(std::vector<stacktrace_frame>& trace, execution_site_t **execution_site, execution_site_t **parent)
{
	//allocation_site_t *allocation_site = root;
	//allocation_site_t *parent = NULL;
	*execution_site = exec_root;
	//std::cerr << "stack begins\n";
	for (auto itr = trace.rbegin(); itr != trace.rend(); ++itr) {
			//std::cerr << "file " << itr->filename << ", line no " << itr->line << "\n";
                        execution_site_t *line = execution_site_table->find(itr->address);
                        if (line == NULL) {
                                execution_site_table->insert(new execution_site_t(
                                                        itr->address, itr->filename, itr->line));
                        }
                        if (exec_root == NULL) {
                                exec_root = new execution_site_t(itr->address);
                                *execution_site = exec_root;

                                *parent = *execution_site;
                                *execution_site = (*execution_site)->get_first_child();
                                continue;
                        }
                        execution_site_t *temp = *execution_site;
                        *execution_site = search_site_at_level(*execution_site, itr->address);
                        if (*execution_site == NULL) {
                                if (temp != NULL) {

                                        while (temp->get_next_sibling() != NULL)
                                                temp = temp->get_next_sibling();
                                        temp->set_next_sibling(new execution_site_t(itr->address));

                                        *execution_site = temp->get_next_sibling();
                                        (*execution_site)->set_parent(temp->get_parent());
                                } else {

                                        (*parent)->set_first_child(new execution_site_t(itr->address));

                                        *execution_site = (*parent)->get_first_child();
                                        (*execution_site)->set_parent(*parent);
                                }
                        }
                        *parent = *execution_site;
                        *execution_site = (*execution_site)->get_first_child();
	}
}
//#endif

void record_exec_context(execution_site_t *parent) {
	if (parent && parent->get_context_id() == 0) {
		parent->set_context_id(++context_counter);
		context_nodes.push_back(new execution_context_t(parent->get_context_id(), parent));
		latest_context = context_counter;
	} else {
		latest_context = parent->get_context_id();
	}
}

void record_object_allocation_context(allocation_site_t *parent) {
	if (parent && parent->get_object_id() == 0) {
		parent->set_object_id(++object_counter);
		object_nodes.push_back(new adm_object_t(parent->get_object_id(), parent, 8));
	} 
}

void log_time(string msg) {
	if (!time_log)
		return;

	std::cout << msg << ": "
		<< std::chrono::time_point_cast<std::chrono::microseconds>(
				std::chrono::steady_clock::now())
		.time_since_epoch()
		.count()
		<< std::endl;
}

int64_t find_nvshmem_dev_of_ptr(int mype, uint64_t mem_addr, int nvshmem_ngpus,
		std::string version) {

	int size = 15;

	int region = -1;

	// 0x000012020000000 is nvshmem's first address for a remote peer
	uint64_t start = 0x000012020000000;

	// 0x000010020000000 is nvshmem's address for the peer itself
	uint64_t incrmnt = (uint64_t)0x000012020000000 - (uint64_t)0x000010020000000;

	for (int i = 1; i <= size; i++) {
		uint64_t bottom = (uint64_t)start + (i - 1) * incrmnt;
		uint64_t top = (uint64_t)start + i * incrmnt;
		if ((uint64_t)bottom <= (uint64_t)mem_addr &&
				(uint64_t)mem_addr < (uint64_t)top) {
			region = i - 1;
			break;
		}
	}

	if (region == -1) {
		return -1;
	}

	if (version == "2.9" || version == "2.8") {
		region += mype;
	}

	if (mype == region) {
		return (mype + 1) % nvshmem_ngpus;
	}

	for (int i = 0; i < size; i++) {
		if (mype == i)
			continue;

		if (region == 0) {
			return i % nvshmem_ngpus;
		}

		region--;
	}

	return -1;
}

uint64_t normalise_nvshmem_ptr(uint64_t mem_addr) {
	return mem_addr & 0x0000F0FFFFFFFFF;
}

int64_t find_dev_of_ptr(uint64_t ptr) {

	for (MemoryAllocation ma : mem_allocs) {
		if (ma.pointer <= ptr && ptr < ma.pointer + ma.bytesize) {
			return ma.deviceID;
		}
	}

	return -1;
}

/* grid launch id, incremented at every launch */
uint64_t grid_launch_id = 0;

const char *whitespace = " ,\"\t\n\r\f\v";

// trim from end of string (right)
inline std::string &rtrim(std::string &s, const char *t = whitespace) {
	s.erase(s.find_last_not_of(t) + 1);
	return s;
}

// trim from beginning of string (left)
inline std::string &ltrim(std::string &s, const char *t = whitespace) {
	s.erase(0, s.find_first_not_of(t));
	return s;
}

// trim from both ends of string (right then left)
inline std::string &trim(std::string &s, const char *t = whitespace) {
	return ltrim(rtrim(s, t), t);
}

void memop_to_line() {
	// open a file in read mode.
	ifstream infile;
	infile.open("memop_to_line.txt");

	if (!infile) {
		cerr << "Please generate a cubin file using nvcc -cubin "
			"-lineinfo command and run nvdisasm --print-line-info "
			"on the generated cubin file with the output directed to "
			"memop_to_line.txt"
			<< endl;
		exit(1);
	}

	int curr_line;
	std::string full_path;
	std::string kern_name;

	for (std::string line; std::getline(infile, line);) {
		std::istringstream input1(line);
		std::string prev_word;
		for (std::string word; std::getline(input1, word, ' ');) {
			if (word.substr(0, 6) == ".text.") {
				rtrim(word, ":");
				kern_name = word;
			}
			if (word == "line" && prev_word.find(".cu") != std::string::npos) {
				full_path = trim(prev_word);
				std::getline(input1, word, ' ');
				curr_line = std::stoi(word);
				get<0>(line_tracking[kern_name]) = full_path;
			}
			if (word.substr(0, 3) == "LDG" || word.substr(0, 3) == "LD.") {

				get<1>(line_tracking[kern_name]).push_back(curr_line);
			} else if (word.substr(0, 3) == "STG" || word.substr(0, 3) == "ST.") {

				get<2>(line_tracking[kern_name]).push_back(curr_line);
			}
			prev_word = word;
		}
	}

	infile.close();
}

std::string find_recorded_kernel(const std::string &curr_kernel) {
	std::string chosen_key;
	size_t shortest_len = 1000;

	for (auto &x : line_tracking) {
		std::string key_str = x.first;

		std::istringstream tokenized_kern_name(curr_kernel);
		std::string name;
		size_t old_pos = 0;
		size_t pos = 0;
		int token_count = 0;
		int match_count = 0;
		while (std::getline(tokenized_kern_name, name, ':')) {
			if (name.length() == 0)
				continue;

			pos = key_str.find(name);
			if (pos != std::string::npos) {

				if (pos >= old_pos) {
					match_count++;
					old_pos = pos;
				}
			}
			token_count++;
		}

		if (token_count != 0 && token_count == match_count &&
				shortest_len > key_str.size()) {
			chosen_key = key_str;
			shortest_len = key_str.size();
		}
	}

	return chosen_key;
}

// Function to print the
// N-ary tree graphically
void printNTree(allocation_site_t *x, vector<bool> flag, int depth = 0,
		bool isLast = false) {

	// Condition when allocation_site is None
	if (x == NULL)
		return;

	// Loop to print the depths of the
	// current allocation_site
	for (int i = 1; i < depth; ++i) {

		// Condition when the depth
		// is exploring
		if (flag[i] == true) {
			cout << "| "
				<< " "
				<< " "
				<< " ";
		}

		// Otherwise print
		// the blank spaces
		else {
			cout << " "
				<< " "
				<< " "
				<< " ";
		}
	}

	// Condition when the current
	// allocation_site is the root allocation_site
	uint64_t pc = x->get_pc();
	int obj_id = x->get_object_id();
	if (depth == 0) {
		cout << pc << endl;

		// Condition when the allocation_site is
		// the last allocation_site of
		// the exploring depth
	} else if (isLast) {
		cout << "+--- " << pc;

		if (obj_id > 0)
			cout << " " << obj_id;

		cout << endl;
		// No more childrens turn it
		// to the non-exploring depth

		flag[depth] = false;

	} else {

		cout << "+--- " << pc;

		if (obj_id > 0)
			cout << " " << obj_id;
		cout << endl;
	}

	x = x->get_first_child();
	// Recursive call for the
	// children allocation_sites

	while (x != NULL) {
		printNTree(x, flag, depth + 1, x->get_next_sibling() == NULL);
		x = x->get_next_sibling();
	}
	flag[depth] = true;
}

void nvbit_at_init() {
	setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);
	GET_VAR_INT(
			instr_begin_interval, "INSTR_BEGIN", 0,
			"Beginning of the instruction interval where to apply instrumentation");
	GET_VAR_INT(instr_end_interval, "INSTR_END", UINT32_MAX,
			"End of the instruction interval where to apply instrumentation");
	GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool");
	GET_VAR_INT(time_log, "TIME_LOG", 0, "Enable time logging inside the tool");
	GET_VAR_INT(
			on_dev_filtering, "ON_DEVICE_FILTERING", 1,
			"Enables on device filtering instead of on host fitering instead ");
	GET_VAR_INT(silent, "SILENT", 0, "Silence long output of the tool");

	GET_VAR_STR(nvshmem_version, "NVSHMEM_VERSION",
			"Specify the nvshmem version to use the correct memory mapping");
	GET_VAR_INT(nvshmem_ngpus, "NVSHMEM_NGPUS", 10,
			"Setting the number of GPUS nvshmem will use");

	GET_VAR_STR(kernel_name, "KERNEL_NAME",
			"Specify the name of the kernel to track");
	GET_VAR_STR(profiled_nccl_file, "PROFILED_NCCL_FILE",
			"Specify the name of the file that has the NCCL function calls");
	GET_VAR_INT(code_attribution, "CODE_ATTRIBUTION", 0,
			"Enable source code line attribution");
	GET_VAR_INT(sample_size, "SAMPLE_SIZE", 1,
			"Setting the sample size, if 100, it means 1/100 of population "
			"is sampled");
	GET_VAR_INT(code_context, "CODE_CONTEXT", 1,
                        "Enable source code line execution context retrieval");
	GET_VAR_INT(data_object_attribution, "DATA_OBJECT_ATTRIBUTION", 1,
                        "Enable data object attribution");

	std::string pad(100, '-');
	if (verbose) {
		std::cout << pad << std::endl;
	}
	// read the file with line info here
	initialize_object_table(100);
	allocation_line_table = new allocation_line_hash_table_t(100);
	initialize_line_table(100);
	execution_site_table = new execution_site_hash_table_t(100);

	if (code_attribution) {
		memop_to_line();
	}
	adm_db_init();
	/* set mutex as recursive */
	string txt_str(".txt");
	pthread_mutexattr_t attr;
	pthread_mutexattr_init(&attr);
	pthread_mutexattr_settype(&attr, PTHREAD_MUTEX_RECURSIVE);
	pthread_mutex_init(&mutex1, &attr);

	if (silent) {
		logger.turnoff();
	}

	log_time("Bgn Snoopie");
}

/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;
std::unordered_map<int, std::string> instrumented_functions;

void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {
	std::string main_func_name(nvbit_get_func_name(ctx, func));

	log_time("Bgn Instrumentation of func: " + main_func_name);
	assert(ctx_state_map.find(ctx) != ctx_state_map.end());
	CTXstate *ctx_state = ctx_state_map[ctx];

	if (already_instrumented.count(func)) {
		log_time("End Instrumentation of func: " + main_func_name);
		return;
	}

	/* Get related functions of the kernel (device function that can be
	 * called by the kernel) */
	std::vector<hipFunction_t> related_functions =
		nvbit_get_related_functions(ctx, func);

	/* add kernel itself to the related function vector */
	related_functions.push_back(func);

	// begin
	if(code_context) {
		std::vector<stacktrace_frame> trace = generate_trace();
		execution_site_t *execution_site = NULL;
		execution_site_t *parent = NULL;
		update_exec_site_tree_cpp(trace, &execution_site, &parent);
		record_exec_context(parent);
	}
	// end

	/* iterate on function */
	for (auto f : related_functions) {
		/* "recording" function was instrumented, if set insertion failed
		 * we have already encountered this function */
		if (!already_instrumented.insert(f).second) {
			continue;
		}

		//int func_id = instrumented_functions.size();
		//instrumented_functions[func_id] = nvbit_get_func_name(ctx, f);

		/* get vector of instructions of function "f" */
		const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);

		if (verbose) {
			std::cout << "instrumenting: " << nvbit_get_func_name(ctx, f)
				<< std::endl;
			printf("MEMTRACE: CTX %p, Inspecting hipFunction_t %p name %s at address "
					"0x%lx\n",
					ctx, f, nvbit_get_func_name(ctx, f), nvbit_get_func_addr(f));
		}

		std::string curr_kernel_name = nvbit_get_func_name(ctx, f);

		//std::cerr << "kernel " << curr_kernel_name << " is launched\n";
		//std::cerr << "call stack:\n";
		//print_trace();
		std::size_t parenthes_pos = curr_kernel_name.find_first_of('(');

		if (parenthes_pos != std::string::npos)
			curr_kernel_name.erase(parenthes_pos);
		std::string encoded_kernel_name;

		std::string file;
		std::string path;

		if (code_attribution) {

			curr_kernel_name = nvbit_get_func_name(ctx, f);
			parenthes_pos = curr_kernel_name.find_first_of('<');
			if (parenthes_pos != std::string::npos) {
				curr_kernel_name.erase(parenthes_pos);
			} else {
				parenthes_pos = curr_kernel_name.find_first_of('(');
				if (parenthes_pos != std::string::npos)
					curr_kernel_name.erase(parenthes_pos);
			}
			std::istringstream tokenized_kern_name(curr_kernel_name);
			std::string name;
			while (std::getline(tokenized_kern_name, name, ' '))
				;
			encoded_kernel_name = find_recorded_kernel(name);
			path = get<0>(line_tracking[encoded_kernel_name]);
			if (path.size() > 0) {
				std::istringstream tokenized_path(path);
				while (std::getline(tokenized_path, file, '/'))
					;
				path.erase(path.size() - file.size() - 1, file.size() + 1);
			}
		}

		// change here
		uint32_t nccl_line_num = 0;
		std::string nccl_filename;
		std::string nccl_dirname;
		if (!profiled_nccl_file.empty()) {
			std::vector<stacktrace_frame> trace = generate_trace();
			allocation_site_t *call_site = root;
			allocation_site_t *parent = NULL;
			for (auto itr = trace.rbegin(); itr != trace.rend(); ++itr) {
				allocation_line_t *line = allocation_line_table->find(itr->address);
				if (line == NULL) {
					allocation_line_table->insert(new allocation_line_t(
								itr->address, itr->symbol, itr->filename, itr->line));
				}
				if (root == NULL) {
					root = new allocation_site_t(itr->address);
					call_site = root;
					parent = call_site;
					call_site = call_site->get_first_child();
					continue;
				}
				allocation_site_t *temp = call_site;
				call_site = search_at_level(call_site, itr->address);
				if (call_site == NULL) {
					if (temp != NULL) {
						while (temp->get_next_sibling() != NULL)
							temp = temp->get_next_sibling();
						temp->set_next_sibling(new allocation_site_t(itr->address));
						call_site = temp->get_next_sibling();
						call_site->set_parent(temp->get_parent());
					} else {
						parent->set_first_child(new allocation_site_t(itr->address));
						call_site = parent->get_first_child();
						call_site->set_parent(parent);
					}
				}
				parent = call_site;
				call_site = call_site->get_first_child();
			}
			string file_name;
			if (parent) {
				file_name =
					allocation_line_table->find(parent->get_pc())->get_file_name();
				while (file_name.find(/*str1*/ profiled_nccl_file) == string::npos) {
					parent = parent->get_parent();
					if (parent)
						file_name =
							allocation_line_table->find(parent->get_pc())->get_file_name();
					else
						break;
				}
			}
			if (parent) {
				allocation_line_t *node = allocation_line_table->find(parent->get_pc());
				path = node->get_file_name();
				if (path.size() > 0) {
					std::istringstream tokenized_path(path);
					while (std::getline(tokenized_path, file, '/'))
						;
					path.erase(path.size() - file.size() - 1, file.size() + 1);
					nccl_line_num = node->get_line_num();
					nccl_filename = file;
					nccl_dirname = path;
				}
			}
		}

		std::string prev_valid_file_name;
		std::string prev_valid_dir_name;
		uint32_t prev_valid_line_num = 0;
		uint32_t cnt = 0;
		int ldg_count = 0;
		int stg_count = 0;
		/* iterate on all the static instructions in the function */
		for (auto instr : instrs) {
			uint32_t instr_offset = instr->getOffset();
			char *file_name = (char *)malloc(sizeof(char) * FILE_NAME_SIZE);
			file_name[0] = '\0';
			char *dir_name = (char *)malloc(sizeof(char) * PATH_NAME_SIZE);
			dir_name[0] = '\0';
			uint32_t line_num = 0;
			bool ret_line_info;
			std::string filename;
			std::string dirname;
			std::string sass;

			if (profiled_nccl_file.empty()) {
				ret_line_info = nvbit_get_line_info(ctx, f, instr_offset, &file_name,
						&dir_name, &line_num);
				filename = file_name;
				dirname = dir_name;
				sass = instr->getSass();
				if (code_attribution && path.size() > 0) {
					std::istringstream input1(sass);
					for (std::string word; std::getline(input1, word, ' ');) {
						if (word.substr(0, 3) == "LDG" || word.substr(0, 3) == "LD.") {
							if (!ret_line_info) {
								line_num = get<1>(line_tracking[encoded_kernel_name])
									[ldg_count]; // line_tracking.first[ldg_count];
								dirname = path;
								filename = file;
							}
							ldg_count++;
						} else if (word.substr(0, 3) == "STG" ||
								word.substr(0, 3) == "ST.") {
							if (!ret_line_info) {
								line_num = get<2>(line_tracking[encoded_kernel_name])
									[stg_count]; // line_tracking.second[stg_count];
								dirname = path;
								filename = file;
							}
							stg_count++;
						}
					}
				}
			} else {
				filename = nccl_filename;
				dirname = nccl_dirname;
				line_num = nccl_line_num;
			}

			short estimated_status = 2; // it is estimated
			if (line_num != 0) {

				estimated_status = 1; // it is original
				adm_line_location_insert(global_index, filename, dirname, sass,
						line_num, estimated_status);
				prev_valid_file_name = filename;
				prev_valid_dir_name = dirname;
				prev_valid_line_num = line_num;
			} else {
				adm_line_location_insert(global_index, prev_valid_file_name,
						prev_valid_dir_name, sass, prev_valid_line_num,
						estimated_status);
			}
			global_index++;
			if (cnt < instr_begin_interval || cnt >= instr_end_interval ||
					instr->getMemorySpace() == InstrType::MemorySpace::NONE ||
					instr->getMemorySpace() == InstrType::MemorySpace::CONSTANT) {
				cnt++;
				continue;
			}
			if (verbose) {
				instr->printDecoded();
			}

			if (opcode_to_id_map.find(instr->getOpcode()) == opcode_to_id_map.end()) {
				int opcode_id = opcode_to_id_map.size();
				opcode_to_id_map[instr->getOpcode()] = opcode_id;
				id_to_opcode_map[opcode_id] = std::string(instr->getOpcode());
			}

			int opcode_id = opcode_to_id_map[instr->getOpcode()];
			int mref_idx = 0;
			/* iterate on the operands */
			for (int i = 0; i < instr->getNumOperands(); i++) {
				/* get the operand "i" */
				const InstrType::operand_t *op = instr->getOperand(i);

				if (op->type == InstrType::OperandType::MREF) {

					/* insert call to the instrumentation function with its
					 * arguments */
					nvbit_insert_call(instr, "instrument_mem", IPOINT_BEFORE);
					/* predicate value */
					nvbit_add_call_arg_guard_pred_val(instr);
					/* opcode id */
					nvbit_add_call_arg_const_val32(instr, opcode_id);
					/* device id */
					int dev_id = -1;
					hipGetDevice(&dev_id);

					nvbit_add_call_arg_const_val32(instr, dev_id);
					//  nvbit_add_call_arg_const_val32(instr, ctx_state->id);
					/* memory reference 64 bit address */
					nvbit_add_call_arg_mref_addr64(instr, mref_idx);
					/* add "space" for kernel function pointer that will be set
					 * at launch time (64 bit value at offset 0 of the dynamic
					 * arguments)*/
					nvbit_add_call_arg_launch_val64(instr, 0);
					/* add pointer to channel_dev*/
					nvbit_add_call_arg_const_val64(instr,
							(uint64_t)ctx_state->channel_dev);
					nvbit_add_call_arg_const_val32(instr, global_index - 1);
					//nvbit_add_call_arg_const_val32(instr, func_id);
					nvbit_add_call_arg_const_val32(instr, latest_context);
					nvbit_add_call_arg_const_val32(instr, sample_size);
					mref_idx++;
				}
			}
			cnt++;
		}
	}

	log_time("End Instrumentation of func: " + main_func_name);
}

__global__ void flush_channel(ChannelDev *ch_dev) {
	/* set a CTA id = -1 to indicate communication thread that this is the
	 * termination flag */
	mem_access_t ma;
	ma.lane_id = -1;
	ch_dev->push(&ma, sizeof(mem_access_t), 0);
	/* flush channel */
	ch_dev->flush();
}

void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
		const char *name, void *params, hipError_t *pStatus) {
	pthread_mutex_lock(&mutex1);
	//std::cerr << "cbid: " << cbid << " " << find_cbid_name(cbid) << " " << API_CUDA_cuLaunchKernelEx << "\n";
	log_time(std::string("Bgn Cuda Event ") + (is_exit ? "Exit" : "Enter") +
			find_cbid_name(cbid));

	/* we prevent re-entry on this callback when issuing CUDA functions inside
	 * this function */
	if (skip_callback_flag || nvshmem_malloc_handled) {
		log_time(std::string("End Cuda Event ") + (is_exit ? "Exit" : "Enter") +
				find_cbid_name(cbid));
		pthread_mutex_unlock(&mutex1);
		return;
	}
	skip_callback_flag = true;

	assert(ctx_state_map.find(ctx) != ctx_state_map.end());
	CTXstate *ctx_state = ctx_state_map[ctx];

	MemoryAllocation ma;
	if (!is_exit && (cbid == API_CUDA_cuLaunchKernel_ptsz ||
				cbid == API_CUDA_cuLaunchKernel)) {
		//fprintf(stderr, "hipModuleLaunchKernel is intercepted\n");
		cuLaunchKernel_params *p = (cuLaunchKernel_params *)params;

		/* Make sure GPU is idle */
		// hipDeviceSynchronize();
		// assert(hipGetLastError() == hipSuccess);

		/* get function name and pc */

		// gets the kernel signature
		std::string func_name(nvbit_get_func_name(ctx, p->f));
		uint64_t pc = nvbit_get_func_addr(p->f);

		std::vector<hipFunction_t> related_functions =
			nvbit_get_related_functions(ctx, p->f);
		related_functions.push_back(p->f);

		for (auto f : related_functions) {

			// NOTE: Needs to verify if cuda_sm_20_div_s64 contains any addrs writes
			// or not. Avoid instrumentting this (possibly a whole family of
			// functions similar to this should be avoided to speed up NCCL
			// profiling)
			if (strcmp(nvbit_get_func_name(ctx, f), "__cuda_sm20_div_s64") == 0) {
				continue;
			}

			// only instrument kernel's with the kernel name supplied by the user,
			// the substr and find are to extract the func name from the func
			// signature
			std::string func_name(nvbit_get_func_name(ctx, f));
			if (kernel_name == "all" ||
					kernel_name == func_name.substr(0, func_name.find("("))) {
				instrument_function_if_needed(ctx, f);
				//std::cerr << "A kernel named " << func_name << " is detected\n";
			} else if (kernel_name == "nccl" &&
					(func_name.substr(0, std::string("ncclKernel").length())
					.compare(std::string("ncclKernel")) == 0 || func_name.substr(0, std::string("ncclDev").length())
                                        .compare(std::string("ncclDev")) == 0)) {
				//std::cerr << "A NCCL kernel is detected 1, name: " << func_name << "\n";
#if 0
				py::object traceback = py::module::import("traceback");
        			py::object extract_summary = traceback.attr("StackSummary").attr("extract");
        			py::object walk_stack = traceback.attr("walk_stack");
        			py::object summary = extract_summary(walk_stack(py::none()));
        			std::vector<py::handle> stack_vec;
        			for (py::handle frame : summary) {
        				std::cerr << frame.attr("filename").attr("__str__")().cast<std::string>() << " " << frame.attr("lineno").attr("__int__")().cast<int>() << " " << frame.attr("name").attr("__str__")().cast<std::string>() << std::endl;
        			}
#endif
				instrument_function_if_needed(ctx, f);
			}

			int nregs = 0;
			CUDA_SAFECALL(hipFuncGetAttribute(&nregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, f));

			int shmem_static_nbytes = 0;
			CUDA_SAFECALL(hipFuncGetAttribute(&shmem_static_nbytes,
						HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, f));

			/* set grid launch id at launch time */
			nvbit_set_at_launch(ctx, f, &grid_launch_id, sizeof(uint64_t));
			/* increment grid launch id for next launch */
			grid_launch_id++;

			/* enable instrumented code to run */
			nvbit_enable_instrumented(ctx, f, true);

			if (verbose) {
				printf("MEMTRACE: CTX 0x%016lx - LAUNCH - Kernel pc 0x%016lx - Kernel "
						"name %s - grid launch id %ld\n",
						(uint64_t)ctx, pc, func_name.c_str(), grid_launch_id);
			}
		}
	} if (!is_exit && cbid == API_CUDA_cuLaunchKernelEx) { 
		//fprintf(stderr, "hipDrvLaunchKernelEx is intercepted\n");

		cuLaunchKernelEx_params *p = (cuLaunchKernelEx_params *)params;

                std::string func_name(nvbit_get_func_name(ctx, p->f));
                uint64_t pc = nvbit_get_func_addr(p->f);

                std::vector<hipFunction_t> related_functions =
                        nvbit_get_related_functions(ctx, p->f);
                related_functions.push_back(p->f);

		for (auto f : related_functions) {

                        // NOTE: Needs to verify if cuda_sm_20_div_s64 contains any addrs writes
                        // or not. Avoid instrumentting this (possibly a whole family of
                        // functions similar to this should be avoided to speed up NCCL
                        // profiling)
                        if (strcmp(nvbit_get_func_name(ctx, f), "__cuda_sm20_div_s64") == 0) {
                                continue;
                        }

                        // only instrument kernel's with the kernel name supplied by the user,
                        // the substr and find are to extract the func name from the func
                        // signature
                        std::string func_name(nvbit_get_func_name(ctx, f));
                        if (kernel_name == "all" ||
                                        kernel_name == func_name.substr(0, func_name.find("("))) {
                                instrument_function_if_needed(ctx, f);
                                //std::cerr << "A kernel named " << func_name << " is detected\n";
                        } else if (kernel_name == "nccl" &&
                                        (func_name.substr(0, std::string("ncclKernel").length())
                                        .compare(std::string("ncclKernel")) == 0 || func_name.substr(0, std::string("ncclDev").length())
                                        .compare(std::string("ncclDev")) == 0)) {
                                //std::cerr << "A NCCL kernel is detected 1, name: " << func_name << "\n";
#if 0
                                py::object traceback = py::module::import("traceback");
                                py::object extract_summary = traceback.attr("StackSummary").attr("extract");
                                py::object walk_stack = traceback.attr("walk_stack");
                                py::object summary = extract_summary(walk_stack(py::none()));
                                std::vector<py::handle> stack_vec;
                                for (py::handle frame : summary) {
                                        std::cerr << frame.attr("filename").attr("__str__")().cast<std::string>() << " " << frame.attr("lineno").attr("__int__")().cast<int>() << " " << frame.attr("name").attr("__str__")().cast<std::string>() << std::endl;
                                }
#endif
                                instrument_function_if_needed(ctx, f);
                        }

			int nregs = 0;
                        CUDA_SAFECALL(hipFuncGetAttribute(&nregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, f));

                        int shmem_static_nbytes = 0;
                        CUDA_SAFECALL(hipFuncGetAttribute(&shmem_static_nbytes,
                                                HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, f));

                        /* set grid launch id at launch time */
                        nvbit_set_at_launch(ctx, f, &grid_launch_id, sizeof(uint64_t));
                        /* increment grid launch id for next launch */
                        grid_launch_id++;

                        /* enable instrumented code to run */
                        nvbit_enable_instrumented(ctx, f, true);

                        if (verbose) {
                                printf("MEMTRACE: CTX 0x%016lx - LAUNCH - Kernel pc 0x%016lx - Kernel "
                                                "name %s - grid launch id %ld\n",
                                                (uint64_t)ctx, pc, func_name.c_str(), grid_launch_id);
                        }
                }

	} else if (!is_exit && (cbid == API_CUDA_cuLaunchCooperativeKernel ||
				cbid == API_CUDA_cuLaunchCooperativeKernel_ptsz)) {
		//fprintf(stderr, "hipModuleLaunchCooperativeKernel is intercepted\n");
		cuLaunchCooperativeKernel_params *p =
			(cuLaunchCooperativeKernel_params *)params;

		/* get function name and pc */
		// gets the kernel signature
		uint64_t pc = nvbit_get_func_addr(p->f);

		std::vector<hipFunction_t> related_functions =
			nvbit_get_related_functions(ctx, p->f);
		related_functions.push_back(p->f);

		// only instrument kernel's with the kernel name supplied by the user,
		// the substr and find are to extract the func name from the func
		// signature
		for (auto f : related_functions) {
			std::string func_name(nvbit_get_func_name(ctx, f));
			if (kernel_name == "all" ||
					kernel_name == func_name.substr(0, func_name.find("("))) {
				/* instrument */
				//std::cerr << "A kernel named " << func_name << " is detected\n";
				instrument_function_if_needed(ctx, p->f);
			} else if (kernel_name == "nccl" &&
					(func_name.substr(0, std::string("ncclKernel").length())
					.compare(std::string("ncclKernel")) == 0 || func_name.substr(0, std::string("ncclDev").length())
                                        .compare(std::string("ncclDev")) == 0)) {
				//std::cerr << "A NCCL kernel is detected, name: " << func_name << "\n";
#if 0
				py::object traceback = py::module::import("traceback");
                                py::object extract_summary = traceback.attr("StackSummary").attr("extract");
                                py::object walk_stack = traceback.attr("walk_stack");
                                py::object summary = extract_summary(walk_stack(py::none()));
                                std::vector<py::handle> stack_vec;
                                for (py::handle frame : summary) {
                                        std::cerr << frame.attr("filename").attr("__str__")().cast<std::string>() << " " << frame.attr("lineno").attr("__int__")().cast<int>() << " " << frame.attr("name").attr("__str__")().cast<std::string>() << std::endl;
                                }
#endif

				instrument_function_if_needed(ctx, f);
			}

			/* set grid launch id at launch time */
			nvbit_set_at_launch(ctx, f, &grid_launch_id, sizeof(uint64_t));
			/* increment grid launch id for next launch */
			grid_launch_id++;

			/* enable instrumented code to run */
			nvbit_enable_instrumented(ctx, f, true);

			if (verbose) {
				printf("MEMTRACE: CTX 0x%016lx - LAUNCH - Kernel pc 0x%016lx - Kernel "
						"name %s - grid launch id %ld\n",
						(uint64_t)ctx, pc, func_name.c_str(), grid_launch_id);
			}
		}
	} else if (is_exit && cbid == API_CUDA_cuMemAlloc_v2) {
		cuMemAlloc_v2_params *p = (cuMemAlloc_v2_params *)params;
		std::stringstream ss;
		ss << HEX(*p->dptr);
		std::stringstream ss2;
		ss2 << HEX(*p->dptr + p->bytesize);
		int deviceID = -1;
		uint64_t pointer = *p->dptr;
		uint64_t bytesize = p->bytesize;

		hipGetDevice(&deviceID);
		assert(hipGetLastError() == hipSuccess);

		ma.deviceID = deviceID;
		ma.pointer = pointer;
		ma.bytesize = bytesize;
		mem_allocs.push_back(ma);

		for (const auto &ctx_map_pair : ctx_state_map) {
			ctx_map_pair.second->channel_dev->add_malloc(ma);
		}
		mem_alloc_count++;

		if (JSON) {
			std::cout << "{\"op\": \"mem_alloc\", "
				<< "\"dev_id\": " << deviceID << ", "
				<< "\"bytesize\": " << p->bytesize << ", \"start\": \""
				<< ss.str() << "\", \"end\": \"" << ss2.str() << "\"}"
				<< std::endl;
		}
	} else if (is_exit && cbid == API_CUDA_cuMemAlloc) {
		cuMemAlloc_params *p = (cuMemAlloc_params *)params;
		std::stringstream ss;
		ss << HEX(*p->dptr);
		std::stringstream ss2;
		ss2 << HEX(*p->dptr + p->bytesize);
		int deviceID = -1;
		uint64_t pointer = *p->dptr;
		uint64_t bytesize = p->bytesize;

		hipGetDevice(&deviceID);
		assert(hipGetLastError() == hipSuccess);

		ma.deviceID = deviceID;
		ma.pointer = pointer;
		ma.bytesize = bytesize;
		mem_allocs.push_back(ma);

		for (const auto &ctx_map_pair : ctx_state_map) {
			ctx_map_pair.second->channel_dev->add_malloc(ma);
		}
		mem_alloc_count++;

		if (JSON) {
			std::cout << "{\"op\": \"mem_alloc\", "
				<< "\"dev_id\": " << deviceID << ", "
				<< "\"bytesize\": " << p->bytesize << ", \"start\": \""
				<< ss.str() << "\", \"end\": \"" << ss2.str() << "\"}"
				<< std::endl;
		}
	} else if (cbid == API_CUDA_cuMemAllocHost) {
		cuMemAllocHost_params *p = (cuMemAllocHost_params *)params;
		std::stringstream ss;
		ss << HEX(*p->pp);
		std::stringstream ss2;
		ss2 << HEX(*p->pp + p->bytesize);
		int deviceID = 999;
		uint64_t pointer = (uint64_t)*p->pp;
		uint64_t bytesize = p->bytesize;
		assert(hipGetLastError() == hipSuccess);

		ma.deviceID = deviceID;
		ma.pointer = pointer;
		ma.bytesize = bytesize;
		mem_allocs.push_back(ma);
		mem_alloc_count++;
		
		for (const auto &ctx_map_pair : ctx_state_map) {
			ctx_map_pair.second->channel_dev->add_malloc(ma);
		}
	}
#if 0 
	else if (cbid == API_CUDA_cuMemAllocHost_v2) {
		//print_trace();
		std::cerr << "API_CUDA_cuMemAllocHost_v2 is detected\n";
		cuMemAllocHost_v2_params *p = (cuMemAllocHost_v2_params *)params;
		std::stringstream ss;
		ss << HEX(*p->pp);
		std::stringstream ss2;
		ss2 << HEX(*p->pp + p->bytesize);
		int deviceID = -1;
		uint64_t pointer = (uint64_t)*p->pp;
		uint64_t bytesize = p->bytesize;
		assert(hipGetLastError() == hipSuccess);

		ma.deviceID = deviceID;
		ma.pointer = pointer;
		ma.bytesize = bytesize;
		mem_allocs.push_back(ma);

		for (const auto &ctx_map_pair : ctx_state_map) {
			ctx_map_pair.second->channel_dev->add_malloc(ma);
		}
	}
#endif 
	else if (cbid == API_CUDA_cuMemHostAlloc) {
		cuMemHostAlloc_params *p = (cuMemHostAlloc_params *)params;
		std::stringstream ss;
		ss << HEX(*p->pp);
		std::stringstream ss2;
		ss2 << HEX(*p->pp + p->bytesize);
		int deviceID = -1;
		uint64_t pointer = (uint64_t)*p->pp;
		uint64_t bytesize = p->bytesize;
		assert(hipGetLastError() == hipSuccess);

		// MemoryAllocation ma = {deviceID, pointer, bytesize};
		ma.deviceID = deviceID;
		ma.pointer = pointer;
		ma.bytesize = bytesize;
		mem_allocs.push_back(ma);

		mem_alloc_count++;
		for (const auto &ctx_map_pair : ctx_state_map) {
			ctx_map_pair.second->channel_dev->add_malloc(ma);
		}
	} else if (is_exit && cbid == API_CUDA_cuMemcpyDtoDAsync_v2) {
		cuMemcpyDtoDAsync_v2_params *p = (cuMemcpyDtoDAsync_v2_params *)params;

		hipDevice_t srcDeviceID;
		hipDevice_t dstDeviceID;

		hipPointerGetAttribute(&srcDeviceID, HIP_POINTER_ATTRIBUTE_DEVICE_ORDINAL,
				p->srcDevice);
		hipPointerGetAttribute(&dstDeviceID, HIP_POINTER_ATTRIBUTE_DEVICE_ORDINAL,
				p->dstDevice);

		adm_range_t *range = nullptr;
		uint64_t offset_address_range = 0;

		if (object_attribution) {
			range = adm_range_find(p->dstDevice);
			offset_address_range = range->get_address();
		}

		// Log this operation

		uint64_t addr1;
		if (p->dstDevice >= 0x0000010020000000) {
			addr1 = normalise_nvshmem_ptr(p->dstDevice);
		} else {
			addr1 = p->dstDevice;
		}
		
		// begin
        	if(code_context) {
                	std::vector<stacktrace_frame> trace = generate_trace();
                	execution_site_t *execution_site = NULL;
                	execution_site_t *parent = NULL;
                	update_exec_site_tree_cpp(trace, &execution_site, &parent);
                	record_exec_context(parent);
        	}
        	// end

		std::stringstream ss;
		ss << find_cbid_name(cbid) << "," << HEX(addr1) << "," << -1 << ","
			<< srcDeviceID << "," << dstDeviceID << "," << -1 << "," << -1 << "," 
			<< ((latest_context > 0) ? latest_context : -1) << ","
			<< -1 << "," << HEX(offset_address_range) << "," << p->ByteCount
			<< std::endl;
		logger.log(ss.str());

	} else if (is_exit && cbid == API_CUDA_cuMemcpyDtoD_v2) {

		// Check if copy operation was successful from the result field

		cuMemcpyDtoD_v2_params *p = (cuMemcpyDtoD_v2_params *)params;
		hipDevice_t srcDeviceID;
		hipDevice_t dstDeviceID;

		hipPointerGetAttribute(&srcDeviceID, HIP_POINTER_ATTRIBUTE_DEVICE_ORDINAL,
				p->srcDevice);
		hipPointerGetAttribute(&dstDeviceID, HIP_POINTER_ATTRIBUTE_DEVICE_ORDINAL,
				p->dstDevice);

		adm_range_t *range = nullptr;
		uint64_t offset_address_range = 0;

		if (object_attribution) {
			range = adm_range_find(p->dstDevice);
			offset_address_range = range->get_address();
		}

		// begin
                if(code_context) {
                        std::vector<stacktrace_frame> trace = generate_trace();
                        execution_site_t *execution_site = NULL;
                        execution_site_t *parent = NULL;
                        update_exec_site_tree_cpp(trace, &execution_site, &parent);
                        record_exec_context(parent);
                }
                // end

		std::stringstream ss;
		ss << find_cbid_name(cbid) << "," << HEX(p->dstDevice) << "," << -1 << ","
                        << srcDeviceID << "," << dstDeviceID << "," << -1 << "," << -1 << "," 
                        << ((latest_context > 0) ? latest_context : -1) << ","
                        << -1 << "," << HEX(offset_address_range) << "," << p->ByteCount
                        << std::endl;

		logger.log(ss.str());
	}

//#if 0
        if(is_exit && cbid == API_CUDA_cuMemMap) {
		//std::cerr << "hipMemMap is detected, memory recorded\n";
                cuMemMap_params *p = (cuMemMap_params *)params;
                std::stringstream ss;
                ss << HEX(p->ptr);
                std::stringstream ss2;
                ss2 << HEX(p->ptr + p->size);
                int deviceID = -1;
                uint64_t pointer = p->ptr;
                uint64_t bytesize = p->size;

                hipGetDevice(&deviceID);
                assert(hipGetLastError() == hipSuccess);

                ma.deviceID = deviceID;
                ma.pointer = pointer;
                ma.bytesize = bytesize;
                mem_allocs.push_back(ma);

		mem_alloc_count++;
                for (const auto &ctx_map_pair : ctx_state_map) {
                        ctx_map_pair.second->channel_dev->add_malloc(ma);
                }

                if (JSON) {
                        std::cout << "{\"op\": \"mem_alloc\", "
                                << "\"dev_id\": " << deviceID << ", "
                                << "\"bytesize\": " << p->size << ", \"start\": \""
                                << ss.str() << "\", \"end\": \"" << ss2.str() << "\"}"
                                << std::endl;
                } 
        }
//#endif
#if 0
	if(is_exit && cbid == API_CUDA_cuMemAllocPitch_v2) {
		std::cerr << "hipMemAllocPitch is detected, call stack begins\n";
		print_trace();
		std::cerr << "hipMemAllocPitch is detected, call stack ends\n";
	}
#endif
	if(is_exit && cbid == API_CUDA_cuMemAllocHost_v2) {
                //std::cerr << "hipMemAllocHost is detected, address recorded\n";
                //print_trace();
		//std::cerr << "API_CUDA_cuMemAllocHost_v2 is detected\n";
                cuMemAllocHost_v2_params *p = (cuMemAllocHost_v2_params *)params;
                std::stringstream ss;
                ss << HEX(*p->pp);
                std::stringstream ss2;
                ss2 << HEX(*p->pp + p->bytesize);
                int deviceID = 999;
                uint64_t pointer = (uint64_t)*p->pp;
                uint64_t bytesize = p->bytesize;
                assert(hipGetLastError() == hipSuccess);

                ma.deviceID = deviceID;
                ma.pointer = pointer;
                ma.bytesize = bytesize;
                mem_allocs.push_back(ma);

		mem_alloc_count++;
                for (const auto &ctx_map_pair : ctx_state_map) {
                        ctx_map_pair.second->channel_dev->add_malloc(ma);
                }
                //std::cerr << "hipMemAllocHost is detected, call stack ends\n";
        }	
	if(is_exit && cbid == API_CUDA_cuMemHostAlloc) {
                //std::cerr << "hipMemAllocHost is detected, memory address range recorded\n";
		//std::cerr << "API_CUDA_cuMemAllocHost_v2 is detected\n";
                cuMemAllocHost_params *p = (cuMemAllocHost_params *)params;
                std::stringstream ss;
                ss << HEX(*p->pp);
                std::stringstream ss2;
                ss2 << HEX(*p->pp + p->bytesize);
                int deviceID = -1;
                uint64_t pointer = (uint64_t)*p->pp;
                uint64_t bytesize = p->bytesize;
                assert(hipGetLastError() == hipSuccess);

                ma.deviceID = deviceID;
                ma.pointer = pointer;
                ma.bytesize = bytesize;
                mem_allocs.push_back(ma);

		mem_alloc_count++;
                for (const auto &ctx_map_pair : ctx_state_map) {
                        ctx_map_pair.second->channel_dev->add_malloc(ma);
                }
                //print_trace();
                //std::cerr << "hipMemAllocHost is detected, call stack ends\n";
        }
#if 0
	if(is_exit && cbid == API_CUDA_cuMemAllocManaged) {
                std::cerr << "hipMallocManaged is detected, call stack begins\n";
                print_trace();
                std::cerr << "hipMallocManaged is detected, call stack ends\n";
        }
	if(is_exit && cbid == API_CUDA_cuArrayCreate_v2) {
                std::cerr << "hipArrayCreate is detected, call stack begins\n";
                print_trace();
                std::cerr << "hipArrayCreate is detected, call stack ends\n";
        }
	if(is_exit && cbid == API_CUDA_cuArray3DCreate_v2) {
                std::cerr << "hipArray3DCreate is detected, call stack begins\n";
                print_trace();
                std::cerr << "hipArray3DCreate is detected, call stack ends\n";
        }
	if(is_exit && cbid == API_CUDA_cuMemAddressReserve) {
                std::cerr << "hipMemAddressReserve is detected, call stack begins\n";
                print_trace();
                std::cerr << "hipMemAddressReserve is detected, call stack ends\n";
        }
	if(is_exit && cbid == API_CUDA_cuMemCreate) {
                std::cerr << "hipMemCreate is detected, call stack begins\n";
                print_trace();
                std::cerr << "hipMemCreate is detected, call stack ends\n";
        }
	if(is_exit && cbid == API_CUDA_cuMemMapArrayAsync_ptsz) {
                std::cerr << "cuMemMapArrayAsync_ptsz is detected, call stack begins\n";
                print_trace();
                std::cerr << "cuMemMapArrayAsync_ptsz is detected, call stack ends\n";
        }
	if(is_exit && cbid == API_CUDA_cuMemAllocAsync_ptsz) {
                std::cerr << "cuMemAllocAsync_ptsz is detected, call stack begins\n";
                print_trace();
                std::cerr << "cuMemAllocAsync_ptsz is detected, call stack ends\n";
        }
	if(is_exit && cbid == API_CUDA_cuMemAllocFromPoolAsync_ptsz) {
                std::cerr << "cuMemAllocFromPoolAsync_ptsz is detected, call stack begins\n";
                print_trace();
                std::cerr << "cuMemAllocFromPoolAsync_ptsz is detected, call stack ends\n";
        }
	if(is_exit && cbid == API_CUDA_cuMemAdvise) {
                std::cerr << "hipMemAdvise is detected, call stack begins\n";
                print_trace();
                std::cerr << "hipMemAdvise is detected, call stack ends\n";
        }
	if(is_exit && cbid == API_CUDA_cuMemAlloc) {
                std::cerr << "hipMalloc is detected, call stack begins\n";
                print_trace();
                std::cerr << "hipMalloc is detected, call stack ends\n";
        }
	if(is_exit && cbid == API_CUDA_cuMemAllocPitch) {
                std::cerr << "hipMemAllocPitch is detected, call stack begins\n";
                print_trace();
                std::cerr << "hipMemAllocPitch is detected, call stack ends\n";
        }
	if(is_exit && cbid == API_CUDA_cuMemGetAddressRange) {
                std::cerr << "hipMemGetAddressRange is detected, call stack begins\n";
                print_trace();
                std::cerr << "hipMemGetAddressRange is detected, call stack ends\n";
        }
	if(is_exit && cbid == API_CUDA_cuMemAllocHost) {
                std::cerr << "hipMemAllocHost is detected, call stack begins\n";
                print_trace();
                std::cerr << "hipMemAllocHost is detected, call stack ends\n";
        }
	if(is_exit && cbid == API_CUDA_cuArrayCreate) {
                std::cerr << "hipArrayCreate is detected, call stack begins\n";
                print_trace();
                std::cerr << "hipArrayCreate is detected, call stack ends\n";
        }
	if(is_exit && cbid == API_CUDA_cuArray3DCreate) {
                std::cerr << "hipArray3DCreate is detected, call stack begins\n";
                print_trace();
                std::cerr << "hipArray3DCreate is detected, call stack ends\n";
        }
	if(is_exit && cbid == API_CUDA_cuMemMapArrayAsync) {
                std::cerr << "hipMemMapArrayAsync is detected, call stack begins\n";
                print_trace();
                std::cerr << "hipMemMapArrayAsync is detected, call stack ends\n";
        }
	if(is_exit && cbid == API_CUDA_cuMemAllocAsync) {
                std::cerr << "hipMemMapArrayAsync is detected, call stack begins\n";
                print_trace();
                std::cerr << "hipMemMapArrayAsync is detected, call stack ends\n";
        }
	if(is_exit && cbid == API_CUDA_cuMemAllocFromPoolAsync) {
                std::cerr << "hipMallocFromPoolAsync is detected, call stack begins\n";
                print_trace();
                std::cerr << "hipMallocFromPoolAsync is detected, call stack ends\n";
        }
#endif

	if (data_object_attribution) {
	if (is_exit &&
			(cbid == API_CUDA_cuMemAlloc || cbid == API_CUDA_cuMemAlloc_v2 ||
			 cbid == API_CUDA_cuMemAllocHost || cbid == API_CUDA_cuMemAllocHost_v2 ||
			 cbid == API_CUDA_cuMemHostAlloc)) {

		std::vector<stacktrace_frame> trace = generate_trace();

		allocation_site_t *allocation_site = root;
		allocation_site_t *parent = NULL;

		for (auto itr = trace.rbegin(); itr != trace.rend(); ++itr) {

			allocation_line_t *line = allocation_line_table->find(itr->address);
			if (line == NULL) {
				allocation_line_table->insert(new allocation_line_t(
							itr->address, itr->symbol, itr->filename, itr->line));
			}
			if (root == NULL) {
				root = new allocation_site_t(itr->address);
				allocation_site = root;

				parent = allocation_site;
				allocation_site = allocation_site->get_first_child();
				continue;
			}
			allocation_site_t *temp = allocation_site;
			allocation_site = search_at_level(allocation_site, itr->address);
			if (allocation_site == NULL) {
				if (temp != NULL) {

					while (temp->get_next_sibling() != NULL)
						temp = temp->get_next_sibling();
					temp->set_next_sibling(new allocation_site_t(itr->address));

					allocation_site = temp->get_next_sibling();
					allocation_site->set_parent(temp->get_parent());
				} else {

					parent->set_first_child(new allocation_site_t(itr->address));

					allocation_site = parent->get_first_child();
					allocation_site->set_parent(parent);
				}
			}
			parent = allocation_site;
			allocation_site = allocation_site->get_first_child();
		}

		string func_name;
		if (parent) {
			func_name =
				allocation_line_table->find(parent->get_pc())->get_func_name();
			while (func_name.find(/*str1*/ "hipMalloc") == string::npos &&
					func_name.find(/*str1*/ "nvshmem_malloc") == string::npos &&
					func_name.find(/*str1*/ "nvshmem_align") == string::npos) {
				parent = parent->get_parent();
				if (parent)
					func_name =
						allocation_line_table->find(parent->get_pc())->get_func_name();
				else
					break;
			}
		}
		if (parent && func_name.find(/*str1*/ "nvshmem_malloc") != string::npos)
			if (parent) {
				while (func_name.find(/*str1*/ "hipMalloc") != string::npos ||
						func_name.find(/*str1*/ "nvshmem_malloc") != string::npos ||
						func_name.find(/*str1*/ "nvshmem_align") != string::npos) {
					parent = parent->get_parent();
					if (parent)
						func_name =
							allocation_line_table->find(parent->get_pc())->get_func_name();
					else
						break;
				}
			}
		if (parent && parent->get_object_id() == 0) {
			parent->set_object_id(++object_counter);
			object_nodes.push_back(
					new adm_object_t(parent->get_object_id(), parent, 8));
		}

		if (parent) {
			adm_range_insert(ma.pointer, ma.bytesize, parent->get_pc(),
					ma.deviceID, "", ADM_STATE_ALLOC);
			range_nodes.push_back(new adm_range_t(
						ma.pointer, ma.bytesize, parent->get_object_id(), ma.deviceID));
		}
	}
	}
	skip_callback_flag = false;
	log_time(std::string("End Cuda Event ") + (is_exit ? "Exit" : "Enter") +
			find_cbid_name(cbid));
	pthread_mutex_unlock(&mutex1);
}

hipError_t cudaMallocHostWrap(void **devPtr, size_t size, const char *var_name,
		const uint32_t element_size, const char *fname,
		const char *fxname, int lineno) {
	hipError_t errorOutput = hipHostMalloc(devPtr, size);
	if (*devPtr) {
		if (!object_attribution) {
			object_attribution = true;
		}
		uint64_t allocation_pc =
			(uint64_t)__builtin_extract_return_addr(__builtin_return_address(0));
		std::string vname = var_name;

		adm_range_t *range = adm_range_find(reinterpret_cast<uint64_t>(*devPtr));
		range->set_var_name(vname);
		if (range) {
			adm_object_t *obj =
				adm_object_insert(allocation_pc, var_name, element_size, fname,
						fxname, lineno, ADM_STATE_ALLOC);
			if (obj) {
				range->set_index_in_object(obj->get_range_count());
				obj->inc_range_count();
			}
		}
	}

	return errorOutput;
}

hipError_t cudaMallocWrap(void **devPtr, size_t size, const char *var_name,
		const uint32_t element_size,
		const char *fname, const char *fxname, int lineno /*, const std::experimental::source_location& location = std::experimental::source_location::current()*/) {
	hipError_t errorOutput = hipMalloc(devPtr, size);
	if (*devPtr) {
		if (!object_attribution) {
			object_attribution = true;
		}
		uint64_t allocation_pc =
			(uint64_t)__builtin_extract_return_addr(__builtin_return_address(0));
		std::string vname = var_name;
		int dev_id = -1;
		hipGetDevice(&dev_id);

		adm_range_t *range = adm_range_find(reinterpret_cast<uint64_t>(*devPtr));
		range->set_var_name(vname);

		if (range) {
			adm_object_t *obj =
				adm_object_insert(allocation_pc, var_name, element_size, fname,
						fxname, lineno, ADM_STATE_ALLOC);
			if (obj) {
				range->set_index_in_object(obj->get_range_count());
				obj->inc_range_count();
			}
		}
	}

	return errorOutput;
}

void *nvshmem_malloc(size_t size) {
	fprintf(stderr, "an nvshmem_malloc is intercepted\n");
	void *(*ori_nvshmem_malloc)(size_t) =
		(void *(*)(size_t))dlsym(RTLD_NEXT, "nvshmem_malloc");
	nvshmem_malloc_handled = true;
	void *allocated_memory = ori_nvshmem_malloc(size);
	fprintf(stderr, "nvshmem_malloc allocates a memory range with offset %lx\n", allocated_memory);
	nvshmem_malloc_handled = false;

	if (data_object_attribution) {
	int deviceID = -1;
	hipGetDevice(&deviceID);
	std::vector<stacktrace_frame> trace = generate_trace();
	allocation_site_t *allocation_site = root;
	allocation_site_t *parent = NULL;
	for (auto itr = trace.rbegin(); itr != trace.rend(); ++itr) {
		allocation_line_t *line = allocation_line_table->find(itr->address);
		if (line == NULL) {
			allocation_line_table->insert(new allocation_line_t(
						itr->address, itr->symbol, itr->filename, itr->line));
		}
		if (root == NULL) {
			root = new allocation_site_t(itr->address);
			allocation_site = root;
			parent = allocation_site;
			allocation_site = allocation_site->get_first_child();
			continue;
		}
		allocation_site_t *temp = allocation_site;
		allocation_site = search_at_level(allocation_site, itr->address);
		if (allocation_site == NULL) {
			if (temp != NULL) {
				while (temp->get_next_sibling() != NULL)
					temp = temp->get_next_sibling();
				temp->set_next_sibling(new allocation_site_t(itr->address));
				allocation_site = temp->get_next_sibling();
				allocation_site->set_parent(temp->get_parent());
			} else {
				parent->set_first_child(new allocation_site_t(itr->address));
				allocation_site = parent->get_first_child();
				allocation_site->set_parent(parent);
			}
		}
		parent = allocation_site;
		allocation_site = allocation_site->get_first_child();
	}

	string func_name;
	if (parent) {
		func_name = allocation_line_table->find(parent->get_pc())->get_func_name();
		while (func_name.find(/*str1*/ "nvshmem_malloc") == string::npos) {
			parent = parent->get_parent();
			if (parent)
				func_name =
					allocation_line_table->find(parent->get_pc())->get_func_name();
			else
				break;
		}
	}

	if (parent) {
		while (func_name.find(/*str1*/ "nvshmem_malloc") != string::npos) {
			parent = parent->get_parent();
			if (parent)
				func_name =
					allocation_line_table->find(parent->get_pc())->get_func_name();
			else
				break;
		}
	}

	if (parent && parent->get_object_id() == 0) {
		parent->set_object_id(++object_counter);
		object_nodes.push_back(
				new adm_object_t(parent->get_object_id(), parent, 8));
	}

	MemoryAllocation ma;
	if (parent) {
		adm_range_insert((uint64_t)allocated_memory, size, parent->get_pc(),
				deviceID, "", ADM_STATE_ALLOC);
		range_nodes.push_back(new adm_range_t((uint64_t)allocated_memory, size,
					parent->get_object_id(), deviceID));
		ma.deviceID = deviceID;
		ma.pointer = (uint64_t)allocated_memory;
		ma.bytesize = size;
		mem_allocs.push_back(ma);
	}
	}
	return allocated_memory;
}

void *nvshmem_alignWrap(
		size_t alignment, size_t size, const char *var_name,
		const uint32_t element_size, const char *fname, const char *fxname, int lineno /*, const std::experimental::source_location& location = std::experimental::source_location::current()*/) {
	void *(*ori_nvshmem_align)(size_t, size_t) =
		(void *(*)(size_t, size_t))dlsym(RTLD_NEXT, "nvshmem_malloc");
	void *allocated_memory = ori_nvshmem_align(alignment, size);
	if (allocated_memory /*&& adm_set_tracing(0)*/) {
		if (!object_attribution) {
			object_attribution = true;
		}
		uint64_t allocation_pc =
			(uint64_t)__builtin_extract_return_addr(__builtin_return_address(0));
		std::string vname = var_name;
		int dev_id = -1;
		hipGetDevice(&dev_id);

		adm_range_t *range =
			adm_range_find(reinterpret_cast<uint64_t>(allocated_memory));
		range->set_var_name(vname);

		if (range) {
			adm_object_t *obj =
				adm_object_insert(allocation_pc, var_name, element_size, fname,
						fxname, lineno, ADM_STATE_ALLOC);
			if (obj) {
				range->set_index_in_object(obj->get_range_count());
				obj->inc_range_count();
			}
		}
	}

	return allocated_memory;
}
//#endif

void *recv_thread_fun(void *args) {

	hipCtx_t ctx = (hipCtx_t)args;

	pthread_mutex_lock(&mutex1);
	/* get context state from map */
	assert(ctx_state_map.find(ctx) != ctx_state_map.end());
	CTXstate *ctx_state = ctx_state_map[ctx];

	int dev_id = -1;
	hipGetDevice(&dev_id);

	log_time(std::string("Bgn Recv Thread " + to_string(dev_id)));

	ChannelHost *ch_host = &ctx_state->channel_host;

	pthread_mutex_unlock(&mutex1);
	char *recv_buffer = (char *)malloc(CHANNEL_SIZE);

	if (!silent && ((int)ctx_state_map.size() - 1 == 0)) {
		std::stringstream ss;
		ss << "op_code, addr, thread_indx, running_dev_id, mem_dev_id, "
			"code_linenum, code_line_index, code_line_context, code_line_estimated_status, "
			"obj_offset, mem_range"
			<< std::endl;
		logger.log(ss.str());
	}

	bool done = false;
	bool waiting = false;
	while (!done) {

		if (!waiting) {
			log_time(std::string("Bgn Waiting Recv Thread " + to_string(dev_id)));
			waiting = true;
		}

		/* receive buffer from channel */
		uint32_t num_recv_bytes = ch_host->recv(recv_buffer, CHANNEL_SIZE);

		if (num_recv_bytes > 0) {
			log_time(std::string("End Waiting Recv Thread " + to_string(dev_id)));
			waiting = false;
			log_time(std::string("Bgn Processing Recv Thread " + to_string(dev_id)));
			uint32_t num_processed_bytes = 0;
			while (num_processed_bytes < num_recv_bytes) {
				mem_access_t *ma = (mem_access_t *)&recv_buffer[num_processed_bytes];

				/* when we receive a CTA_id_x it means all the kernels
				 * completed, this is the special token we receive from the
				 * flush channel kernel that is issues at the end of the
				 * context */

				if (ma->lane_id == -1) {
					done = true;
					break;
				}

				adm_range_t *range = nullptr; // adm_range_find(ma.addrs[0]);
				uint64_t allocation_pc = 0;   // obj->get_allocation_pc();
				std::string varname;
				std::string filename;
				std::string funcname;
				uint32_t linenum;
				uint32_t data_type_size = 1;
				int dev_id = -1;
				int line_index = ma->global_index;
				std::string line_filename = get_line_file_name(line_index);
				std::string line_dirname = get_line_dir_name(line_index);
				std::string line_sass = get_line_sass(line_index);
				uint32_t line_linenum = get_line_line_num(line_index);
				short line_estimated_status = get_line_estimated_status(line_index);
				uint64_t offset_address_range = 0;

				for (int i = 0; i < 32; i++) {

					if (ma->addrs[i] == 0x0)
						continue;

					int mem_device_id = -1;
					if(on_dev_filtering) {
						mem_device_id = ma->owner_id;
					} else {
						mem_device_id = find_dev_of_ptr(ma->addrs[i]);
					}

					// nvshmem heap_base = 0x10020000000
					// ignore operations on memory locations not allocated by hipMalloc
					// on the host
					bool nvshmem_flag = false;
					if (mem_device_id == -1 && (ma->addrs[i] >= 0x0000010020000000)) {
						nvshmem_flag = true;
						mem_device_id = find_nvshmem_dev_of_ptr(
								ma->dev_id, ma->addrs[i], nvshmem_ngpus, nvshmem_version);
					}

					// ignore operations on the same device
					if (mem_device_id == ma->dev_id)
						continue;

					if (mem_device_id == -1)
						continue;

					uint32_t index_in_object = 0;
					uint32_t index_in_malloc = 0;

					if (silent)
						continue;

					std::stringstream ss;
					uint64_t addr1;
					if (nvshmem_flag) {
						addr1 = normalise_nvshmem_ptr(ma->addrs[i]);
					} else {
						addr1 = ma->addrs[i];
					}

					range = adm_range_find(addr1);
					if (range != nullptr) {
						allocation_pc = range->get_allocation_pc();
						if (object_exists(allocation_pc)) {
							varname = get_object_var_name(allocation_pc);
							filename = get_object_file_name(allocation_pc);
							funcname = get_object_func_name(allocation_pc);
							linenum = get_object_line_num(allocation_pc);
							dev_id = range->get_device_id();
							data_type_size = get_object_data_type_size(allocation_pc);
							index_in_object = range->get_index_in_object();
						}
						index_in_malloc =
							(ma->addrs[i] - range->get_address()) / data_type_size;
						offset_address_range = range->get_address();
					}

					if (JSON) {
						ss << "{\"op\": \"" << id_to_opcode_map[ma->opcode_id] << "\", "
							//<< "\"kernel_name\": \"" << instrumented_functions[ma->func_id]
							<< "\", "
							<< "\"addr\": \"" << HEX(addr1) << "\","
							<< "\"object_allocation_pc\": \"" << HEX(allocation_pc) << "\", "
							<< "\"object_variable_name\": \"" << varname << "\", "
							<< "\"malloc_index_in_object\": " << index_in_object << ", "
							<< "\"element_index_in_malloc\": " << index_in_malloc << ", "
							<< "\"object_allocation_file_name\": \"" << filename << "\", "
							<< "\"object_allocation_func_name\": \"" << funcname << "\", "
							<< "\"object_allocation_line_num\": " << linenum << ", "
							<< "\"object_allocation_device_id\": " << dev_id << ", "
							<< "\"thread_index\": " << ma->thread_index << ", "
							<< "\"lane_id\": " << ma->lane_id << ", "
							<< "\"running_device_id\": " << ma->dev_id << ", "
							<< "\"mem_device_id\": " << mem_device_id << ", "
							<< "\"code_line_index\": \"" << line_index << "\", "
							<< "\"code_line_filename\": \"" << line_filename << "\", "
							<< "\"code_line_dirname\": \"" << line_dirname << "\", "
							<< "\"code_line_linenum\": " << line_linenum << ", "
							<< "\"code_line_estimated_status\": " << line_estimated_status
							<< "}" << std::endl;
					} else {
						ss << id_to_opcode_map[ma->opcode_id] << "," << HEX(addr1) << ","
							<< ma->thread_index << "," << ma->dev_id << "," << mem_device_id
							<< "," << line_linenum << "," << line_index << "," << ma->context_id << ","
							<< line_estimated_status << "," << HEX(offset_address_range)
							<< "," << 4 << std::endl;
					}
					logger.log(ss.str());
					// memop_outfile << ss.str() << std::flush;
				}
				num_processed_bytes += sizeof(mem_access_t);
			}

			log_time(std::string("End Processing Recv Thread " + to_string(dev_id)));
		}
	}

	log_time(std::string("End Recv Thread " + to_string(dev_id)));
	return NULL;
}

void nvbit_at_ctx_init(hipCtx_t ctx) {
	pthread_mutex_lock(&mutex1);
	int dev_id = -1;
	hipGetDevice(&dev_id);

	log_time("Bgn Context" + to_string(dev_id));
	if (verbose) {
		printf("MEMTRACE: STARTING CONTEXT %p\n", ctx);
	}
	CTXstate *ctx_state = new CTXstate;
	assert(ctx_state_map.find(ctx) == ctx_state_map.end());
	ctx_state_map[ctx] = ctx_state;
	hipMallocManaged(&ctx_state->channel_dev, sizeof(ChannelDev));

	if(on_dev_filtering)
		std::cerr << "on_dev_filtering is active";
	ctx_state->channel_host.init((int)ctx_state_map.size() - 1, CHANNEL_SIZE,
			ctx_state->channel_dev, recv_thread_fun,
			on_dev_filtering, ctx);
	nvbit_set_tool_pthread(ctx_state->channel_host.get_thread());
	pthread_mutex_unlock(&mutex1);
}

void nvbit_at_ctx_term(hipCtx_t ctx) {
	pthread_mutex_lock(&mutex1);
	int dev_id = -1;
	hipGetDevice(&dev_id);
	log_time("End Context" + to_string(dev_id));

	skip_callback_flag = true;
	if (verbose) {
		printf("MEMTRACE: TERMINATING CONTEXT %p\n", ctx);
	}
	/* get context state from map */
	assert(ctx_state_map.find(ctx) != ctx_state_map.end());
	CTXstate *ctx_state = ctx_state_map[ctx];

	/* flush channel */
	flush_channel<<<1, 1>>>(ctx_state->channel_dev);
	/* Make sure flush of channel is complete */
	hipDeviceSynchronize();
	assert(hipGetLastError() == hipSuccess);

	ctx_state->channel_host.destroy(false);
	hipFree(ctx_state->channel_dev);
	skip_callback_flag = false;
	delete ctx_state;
	pthread_mutex_unlock(&mutex1);
}

void nvbit_at_term() {
	std::cerr << "Number of detected memory allocations: " << mem_alloc_count << "\n";
	if (silent) {
		return;
	}

	if (object_attribution) {
		adm_ranges_print();
	}
	if(code_attribution)
		adm_line_table_print();

	if(data_object_attribution) {
		ofstream object_outfile;
		string object_str("mem_alloc_site_log_");
		string txt_str(".txt");
		string object_log_str = object_str + to_string(getpid()) + txt_str;
		object_outfile.open(object_log_str);
		object_outfile << "pc,func_name,file_name,line_no\n";
		allocation_line_table->print(object_outfile);
		object_outfile.close();

		ofstream object_outfile1;
		string object_str1("address_range_log_");
		string object_log_str1 = object_str1 + to_string(getpid()) + txt_str;
		object_outfile1.open(object_log_str1);
		object_outfile1 << "offset,size,obj_id,dev_id\n";
		for (auto i : range_nodes)
			i->print(object_outfile1);
		object_outfile1.close();

		ofstream object_outfile2;
		string object_str2("data_object_log_");
		string object_log_str2 = object_str2 + to_string(getpid()) + txt_str;
		object_outfile2.open(object_log_str2);
		object_outfile2 << "obj_id,var_name,call_stack\n";
		for (auto i : object_nodes)
			i->print(object_outfile2);
		object_outfile2.close();
	}

	if(code_context) {
		ofstream object_outfile3;
        	string object_str3("exec_site_log_");
		string txt_str(".txt");
        	string object_log_str3 = object_str3 + to_string(getpid()) + txt_str;
        	object_outfile3.open(object_log_str3);
        	object_outfile3 << "site_id,file,code_linenum\n";
        	execution_site_table->print(object_outfile3);
        	object_outfile3.close();

		ofstream object_outfile4;
        	string object_str4("exec_context_log_");
        	string object_log_str4 = object_str4 + to_string(getpid()) + txt_str;
        	object_outfile4.open(object_log_str4);
        	object_outfile4 << "context_id,call_stack\n";
        	for (auto i : context_nodes)
                	i->print(object_outfile4);
        	object_outfile4.close();	
	}

	object_nodes.clear();
	context_nodes.clear();
	delete allocation_line_table;
	delete execution_site_table;
	delete root;
	log_time("End Snoopie");
	adm_db_fini();
}
