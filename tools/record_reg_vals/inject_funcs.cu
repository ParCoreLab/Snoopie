#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdint.h>
#include <stdio.h>
#include <cstdarg>

#include "utils/utils.h"

/* for channel */
#include "utils/channel.hpp"

/* contains definition of the mem_access_t structure */
#include "common.h"

extern "C" __device__ __noinline__ void record_reg_val(int pred, int opcode_id,
                                                       uint64_t pchannel_dev,
                                                       int32_t num_regs...) {
    if (!pred) {
        return;
    }

    int active_mask = __ballot_sync(__activemask(), 1);
    const int laneid = get_laneid();
    const int first_laneid = __ffs(active_mask) - 1;

    reg_info_t ri;

    int4 cta = get_ctaid();
    ri.cta_id_x = cta.x;
    ri.cta_id_y = cta.y;
    ri.cta_id_z = cta.z;
    ri.warp_id = get_warpid();
    ri.opcode_id = opcode_id;
    ri.num_regs = num_regs;

    if (num_regs) {
        va_list vl;
        va_start(vl, num_regs);

        for (int i = 0; i < num_regs; i++) {
            uint32_t val = va_arg(vl, uint32_t);

            /* collect register values from other threads */
            for (int tid = 0; tid < 32; tid++) {
                ri.reg_vals[tid][i] = __shfl_sync(active_mask, val, tid);
            }
        }
        va_end(vl);
    }

    /* first active lane pushes information on the channel */
    if (first_laneid == laneid) {
        ChannelDev *channel_dev = (ChannelDev *)pchannel_dev;
        channel_dev->push(&ri, sizeof(reg_info_t));
    }
}
