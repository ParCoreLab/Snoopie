#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <unistd.h>
#include <string>
#include <map>
#include <vector>
#include <unordered_set>

/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

/* for channel */
#include "utils/channel.hpp"

/* contains definition of the reg_info_t structure */
#include "common.h"

/* Channel used to communicate from GPU to CPU receiving thread */
#define CHANNEL_SIZE (1l << 20)
static __managed__ ChannelDev channel_dev;
static ChannelHost channel_host;

/* receiving thread and its control variables */
pthread_t recv_thread;
volatile bool recv_thread_started = false;
volatile bool recv_thread_receiving = false;

/* skip flag used to avoid re-entry on the nvbit_callback when issuing
 * flush_channel kernel call */
bool skip_flag = false;

/* global control variables for this tool */
uint32_t instr_begin_interval = 0;
uint32_t instr_end_interval = UINT32_MAX;
int verbose = 0;

/* opcode to id map and reverse map  */
std::map<std::string, int> sass_to_id_map;
std::map<int, std::string> id_to_sass_map;

void nvbit_at_init() {
    setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);
    GET_VAR_INT(
        instr_begin_interval, "INSTR_BEGIN", 0,
        "Beginning of the instruction interval where to apply instrumentation");
    GET_VAR_INT(
        instr_end_interval, "INSTR_END", UINT32_MAX,
        "End of the instruction interval where to apply instrumentation");
    GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool");
    std::string pad(100, '-');
    printf("%s\n", pad.c_str());
}
/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;

void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {
    /* Get related functions of the kernel (device function that can be
     * called by the kernel) */
    std::vector<hipFunction_t> related_functions =
        nvbit_get_related_functions(ctx, func);

    /* add kernel itself to the related function vector */
    related_functions.push_back(func);

    /* iterate on function */
    for (auto f : related_functions) {
        /* "recording" function was instrumented, if set insertion failed
         * we have already encountered this function */
        if (!already_instrumented.insert(f).second) {
            continue;
        }
        const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);
        if (verbose) {
            printf("Inspecting function %s at address 0x%lx\n",
                   nvbit_get_func_name(ctx, f), nvbit_get_func_addr(f));
        }

        uint32_t cnt = 0;
        /* iterate on all the static instructions in the function */
        for (auto instr : instrs) {
            if (cnt < instr_begin_interval || cnt >= instr_end_interval) {
                cnt++;
                continue;
            }
            if (verbose) {
                instr->printDecoded();
            }

            if (sass_to_id_map.find(instr->getSass()) ==
                sass_to_id_map.end()) {
                int opcode_id = sass_to_id_map.size();
                sass_to_id_map[instr->getSass()] = opcode_id;
                id_to_sass_map[opcode_id] = std::string(instr->getSass());
            }

            int opcode_id = sass_to_id_map[instr->getSass()];
            std::vector<int> reg_num_list;
            /* iterate on the operands */
            for (int i = 0; i < instr->getNumOperands(); i++) {
                /* get the operand "i" */
                const InstrType::operand_t *op = instr->getOperand(i);
                if (op->type == InstrType::OperandType::REG) {
                    for (int reg_idx = 0; reg_idx < instr->getSize() / 4; reg_idx++) {
                        reg_num_list.push_back(op->u.reg.num + reg_idx);
                    }
                }
            }
            /* insert call to the instrumentation function with its
             * arguments */
            nvbit_insert_call(instr, "record_reg_val", IPOINT_BEFORE);
            /* guard predicate value */
            nvbit_add_call_arg_guard_pred_val(instr);
            /* opcode id */
            nvbit_add_call_arg_const_val32(instr, opcode_id);
            /* add pointer to channel_dev*/
            nvbit_add_call_arg_const_val64(instr,
                                           (uint64_t)&channel_dev);
            /* how many register values are passed next */
            nvbit_add_call_arg_const_val32(instr, reg_num_list.size());
            for (int num : reg_num_list) {
                /* last parameter tells it is a variadic parameter passed to
                 * the instrument function record_reg_val() */
                nvbit_add_call_arg_reg_val(instr, num, true);
            }
            cnt++;
        }
    }
}

__global__ void flush_channel() {
    /* push memory access with negative cta id to communicate the kernel is
     * completed */
    reg_info_t ri;
    ri.cta_id_x = -1;
    channel_dev.push(&ri, sizeof(reg_info_t));

    /* flush channel */
    channel_dev.flush();
}

void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char *name, void *params, hipError_t *pStatus) {
    if (skip_flag) return;

    if (cbid == API_CUDA_cuLaunchKernel_ptsz ||
        cbid == API_CUDA_cuLaunchKernel) {
        cuLaunchKernel_params *p = (cuLaunchKernel_params *)params;

        if (!is_exit) {
            int nregs;
            CUDA_SAFECALL(
                hipFuncGetAttribute(&nregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, p->f));

            int shmem_static_nbytes;
            CUDA_SAFECALL(
                hipFuncGetAttribute(&shmem_static_nbytes,
                                   HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, p->f));

            instrument_function_if_needed(ctx, p->f);

            nvbit_enable_instrumented(ctx, p->f, true);

            printf(
                "Kernel %s - grid size %d,%d,%d - block size %d,%d,%d - nregs "
                "%d - shmem %d - cuda stream id %ld\n",
                nvbit_get_func_name(ctx, p->f), p->gridDimX, p->gridDimY,
                p->gridDimZ, p->blockDimX, p->blockDimY, p->blockDimZ, nregs,
                shmem_static_nbytes + p->sharedMemBytes, (uint64_t)p->hStream);
            recv_thread_receiving = true;

        } else {
            /* make sure current kernel is completed */
            hipDeviceSynchronize();
            hipError_t kernelError = hipGetLastError();
            if (kernelError != hipSuccess) {
                printf("Kernel launch error: %s\n", hipGetErrorString(kernelError));
                assert(0);
            }

            /* make sure we prevent re-entry on the nvbit_callback when issuing
             * the flush_channel kernel */
            skip_flag = true;

            /* issue flush of channel so we are sure all the memory accesses
             * have been pushed */
            flush_channel<<<1, 1>>>();
            hipDeviceSynchronize();
            assert(hipGetLastError() == hipSuccess);

            /* unset the skip flag */
            skip_flag = false;

            /* wait here until the receiving thread has not finished with the
             * current kernel */
            while (recv_thread_receiving) {
                pthread_yield();
            }
        }
    }
}

void *recv_thread_fun(void *) {
    char *recv_buffer = (char *)malloc(CHANNEL_SIZE);

    while (recv_thread_started) {
        uint32_t num_recv_bytes = 0;
        if (recv_thread_receiving &&
            (num_recv_bytes = channel_host.recv(recv_buffer, CHANNEL_SIZE)) >
                0) {
            uint32_t num_processed_bytes = 0;
            while (num_processed_bytes < num_recv_bytes) {
                reg_info_t *ri =
                    (reg_info_t *)&recv_buffer[num_processed_bytes];

                /* when we get this cta_id_x it means the kernel has completed
                 */
                if (ri->cta_id_x == -1) {
                    recv_thread_receiving = false;
                    break;
                }

                printf("CTA %d,%d,%d - warp %d - %s:\n", ri->cta_id_x,
                       ri->cta_id_y, ri->cta_id_z, ri->warp_id,
                       id_to_sass_map[ri->opcode_id].c_str());

                for (int reg_idx = 0; reg_idx < ri->num_regs; reg_idx++) {
                    printf("* ");
                    for (int i = 0; i < 32; i++) {
                        printf("Reg%d_T%d: 0x%08x ", reg_idx, i,
                               ri->reg_vals[i][reg_idx]);
                    }
                    printf("\n");
                }

                printf("\n");
                num_processed_bytes += sizeof(reg_info_t);
            }
        }
    }
    free(recv_buffer);
    return NULL;
}

void nvbit_at_ctx_init(hipCtx_t ctx) {
    recv_thread_started = true;
    channel_host.init(0, CHANNEL_SIZE, &channel_dev, NULL);
    pthread_create(&recv_thread, NULL, recv_thread_fun, NULL);
}

void nvbit_at_ctx_term(hipCtx_t ctx) {
    if (recv_thread_started) {
        recv_thread_started = false;
        pthread_join(recv_thread, NULL);
    }
}
