#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <unistd.h>
#include <map>
#include <sstream>
#include <string>
#include <unordered_set>
#include <unordered_map>

/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

/* for channel */
#include "utils/channel.hpp"

/* contains definition of the mem_access_t structure */
#include "common.h"

#define HEX(x)                                                            \
    "0x" << std::setfill('0') << std::setw(16) << std::hex << (uint64_t)x \
         << std::dec

#define CHANNEL_SIZE (1l << 20)

struct CTXstate {
    /* context id */
    int id;

    /* Channel used to communicate from GPU to CPU receiving thread */
    ChannelDev* channel_dev;
    ChannelHost channel_host;
};

/* lock */
pthread_mutex_t mutex;

/* map to store context state */
std::unordered_map<hipCtx_t, CTXstate*> ctx_state_map;

/* skip flag used to avoid re-entry on the nvbit_callback when issuing
 * flush_channel kernel call */
bool skip_callback_flag = false;

/* global control variables for this tool */
uint32_t instr_begin_interval = 0;
uint32_t instr_end_interval = UINT32_MAX;
int verbose = 0;

/* opcode to id map and reverse map  */
std::map<std::string, int> opcode_to_id_map;
std::map<int, std::string> id_to_opcode_map;

/* grid launch id, incremented at every launch */
uint64_t grid_launch_id = 0;

void nvbit_at_init() {
    setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);
    GET_VAR_INT(
        instr_begin_interval, "INSTR_BEGIN", 0,
        "Beginning of the instruction interval where to apply instrumentation");
    GET_VAR_INT(
        instr_end_interval, "INSTR_END", UINT32_MAX,
        "End of the instruction interval where to apply instrumentation");
    GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool");
    std::string pad(100, '-');
    printf("%s\n", pad.c_str());

    /* set mutex as recursive */
    pthread_mutexattr_t attr;
    pthread_mutexattr_init(&attr);
    pthread_mutexattr_settype(&attr, PTHREAD_MUTEX_RECURSIVE);
    pthread_mutex_init(&mutex, &attr);
}

/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;

void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {
    assert(ctx_state_map.find(ctx) != ctx_state_map.end());
    CTXstate* ctx_state = ctx_state_map[ctx];

    /* Get related functions of the kernel (device function that can be
     * called by the kernel) */
    std::vector<hipFunction_t> related_functions =
        nvbit_get_related_functions(ctx, func);

    /* add kernel itself to the related function vector */
    related_functions.push_back(func);

    /* iterate on function */
    for (auto f : related_functions) {
        /* "recording" function was instrumented, if set insertion failed
         * we have already encountered this function */
        if (!already_instrumented.insert(f).second) {
            continue;
        }

        /* get vector of instructions of function "f" */
        const std::vector<Instr*>& instrs = nvbit_get_instrs(ctx, f);

        if (verbose) {
            printf(
                "MEMTRACE: CTX %p, Inspecting hipFunction_t %p name %s at address "
                "0x%lx\n",
                ctx, f, nvbit_get_func_name(ctx, f), nvbit_get_func_addr(f));
        }

        uint32_t cnt = 0;
        /* iterate on all the static instructions in the function */
        for (auto instr : instrs) {
            if (cnt < instr_begin_interval || cnt >= instr_end_interval ||
                instr->getMemorySpace() == InstrType::MemorySpace::NONE ||
                instr->getMemorySpace() == InstrType::MemorySpace::CONSTANT) {
                cnt++;
                continue;
            }
            if (verbose) {
                instr->printDecoded();
            }

            if (opcode_to_id_map.find(instr->getOpcode()) ==
                opcode_to_id_map.end()) {
                int opcode_id = opcode_to_id_map.size();
                opcode_to_id_map[instr->getOpcode()] = opcode_id;
                id_to_opcode_map[opcode_id] = std::string(instr->getOpcode());
            }

            int opcode_id = opcode_to_id_map[instr->getOpcode()];
            int mref_idx = 0;
            /* iterate on the operands */
            for (int i = 0; i < instr->getNumOperands(); i++) {
                /* get the operand "i" */
                const InstrType::operand_t* op = instr->getOperand(i);

                if (op->type == InstrType::OperandType::MREF) {
                    /* insert call to the instrumentation function with its
                     * arguments */
                    nvbit_insert_call(instr, "instrument_mem", IPOINT_BEFORE);
                    /* predicate value */
                    nvbit_add_call_arg_guard_pred_val(instr);
                    /* opcode id */
                    nvbit_add_call_arg_const_val32(instr, opcode_id);
                    /* memory reference 64 bit address */
                    nvbit_add_call_arg_mref_addr64(instr, mref_idx);
                    /* add "space" for kernel function pointer that will be set
                     * at launch time (64 bit value at offset 0 of the dynamic
                     * arguments)*/
                    nvbit_add_call_arg_launch_val64(instr, 0);
                    /* add pointer to channel_dev*/
                    nvbit_add_call_arg_const_val64(
                        instr, (uint64_t)ctx_state->channel_dev);
                    mref_idx++;
                }
            }
            cnt++;
        }
    }
}

__global__ void flush_channel(ChannelDev* ch_dev) {
    /* set a CTA id = -1 to indicate communication thread that this is the
     * termination flag */
    mem_access_t ma;
    ma.cta_id_x = -1;
    ch_dev->push(&ma, sizeof(mem_access_t));
    /* flush channel */
    ch_dev->flush();
}

void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char* name, void* params, hipError_t* pStatus) {
    pthread_mutex_lock(&mutex);

    /* we prevent re-entry on this callback when issuing CUDA functions inside
     * this function */
    if (skip_callback_flag) {
        pthread_mutex_unlock(&mutex);
        return;
    }
    skip_callback_flag = true;

    assert(ctx_state_map.find(ctx) != ctx_state_map.end());
    CTXstate* ctx_state = ctx_state_map[ctx];

    if (cbid == API_CUDA_cuLaunchKernel_ptsz ||
        cbid == API_CUDA_cuLaunchKernel) {
        cuLaunchKernel_params* p = (cuLaunchKernel_params*)params;

        /* Make sure GPU is idle */
        hipDeviceSynchronize();
        assert(hipGetLastError() == hipSuccess);

        if (!is_exit) {
            /* instrument */
            instrument_function_if_needed(ctx, p->f);

            int nregs = 0;
            CUDA_SAFECALL(
                hipFuncGetAttribute(&nregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, p->f));

            int shmem_static_nbytes = 0;
            CUDA_SAFECALL(
                hipFuncGetAttribute(&shmem_static_nbytes,
                                   HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, p->f));

            /* get function name and pc */
            const char* func_name = nvbit_get_func_name(ctx, p->f);
            uint64_t pc = nvbit_get_func_addr(p->f);

            /* set grid launch id at launch time */
            nvbit_set_at_launch(ctx, p->f, &grid_launch_id, sizeof(uint64_t));
            /* increment grid launch id for next launch */
            grid_launch_id++;

            /* enable instrumented code to run */
            nvbit_enable_instrumented(ctx, p->f, true);

            printf(
                "MEMTRACE: CTX 0x%016lx - LAUNCH - Kernel pc 0x%016lx - Kernel "
                "name %s - grid launch id %ld - grid size %d,%d,%d - block "
                "size %d,%d,%d - nregs %d - shmem %d - cuda stream id %ld\n",
                (uint64_t)ctx, pc, func_name, grid_launch_id, p->gridDimX,
                p->gridDimY, p->gridDimZ, p->blockDimX, p->blockDimY,
                p->blockDimZ, nregs, shmem_static_nbytes + p->sharedMemBytes,
                (uint64_t)p->hStream);
        }
    }
    skip_callback_flag = false;
    pthread_mutex_unlock(&mutex);
}

void* recv_thread_fun(void* args) {
    hipCtx_t ctx = (hipCtx_t)args;

    pthread_mutex_lock(&mutex);
    /* get context state from map */
    assert(ctx_state_map.find(ctx) != ctx_state_map.end());
    CTXstate* ctx_state = ctx_state_map[ctx];

    ChannelHost* ch_host = &ctx_state->channel_host;
    pthread_mutex_unlock(&mutex);
    char* recv_buffer = (char*)malloc(CHANNEL_SIZE);

    bool done = false;
    while (!done) {
        /* receive buffer from channel */
        uint32_t num_recv_bytes = ch_host->recv(recv_buffer, CHANNEL_SIZE);
        if (num_recv_bytes > 0) {
            uint32_t num_processed_bytes = 0;
            while (num_processed_bytes < num_recv_bytes) {
                mem_access_t* ma =
                    (mem_access_t*)&recv_buffer[num_processed_bytes];

                /* when we receive a CTA_id_x it means all the kernels
                 * completed, this is the special token we receive from the
                 * flush channel kernel that is issues at the end of the
                 * context */
                if (ma->cta_id_x == -1) {
                    done = true;
                    break;
                }

                std::stringstream ss;
                ss << "CTX " << HEX(ctx) << " - grid_launch_id "
                   << ma->grid_launch_id << " - CTA " << ma->cta_id_x << ","
                   << ma->cta_id_y << "," << ma->cta_id_z << " - warp "
                   << ma->warp_id << " - " << id_to_opcode_map[ma->opcode_id]
                   << " - ";

                for (int i = 0; i < 32; i++) {
                    ss << HEX(ma->addrs[i]) << " ";
                }

                printf("MEMTRACE: %s\n", ss.str().c_str());
                num_processed_bytes += sizeof(mem_access_t);
            }
        }
    }
    free(recv_buffer);
    return NULL;
}

void nvbit_at_ctx_init(hipCtx_t ctx) {
    pthread_mutex_lock(&mutex);
    if (verbose) {
        printf("MEMTRACE: STARTING CONTEXT %p\n", ctx);
    }
    CTXstate* ctx_state = new CTXstate;
    assert(ctx_state_map.find(ctx) == ctx_state_map.end());
    ctx_state_map[ctx] = ctx_state;
    hipMallocManaged(&ctx_state->channel_dev, sizeof(ChannelDev));
    ctx_state->channel_host.init((int)ctx_state_map.size() - 1, CHANNEL_SIZE,
                                 ctx_state->channel_dev, recv_thread_fun, ctx);
    nvbit_set_tool_pthread(ctx_state->channel_host.get_thread());
    pthread_mutex_unlock(&mutex);
}

void nvbit_at_ctx_term(hipCtx_t ctx) {
    pthread_mutex_lock(&mutex);
    skip_callback_flag = true;
    if (verbose) {
        printf("MEMTRACE: TERMINATING CONTEXT %p\n", ctx);
    }
    /* get context state from map */
    assert(ctx_state_map.find(ctx) != ctx_state_map.end());
    CTXstate* ctx_state = ctx_state_map[ctx];

    /* flush channel */
    flush_channel<<<1, 1>>>(ctx_state->channel_dev);
    /* Make sure flush of channel is complete */
    hipDeviceSynchronize();
    assert(hipGetLastError() == hipSuccess);

    ctx_state->channel_host.destroy(false);
    hipFree(ctx_state->channel_dev);
    skip_callback_flag = false;
    delete ctx_state;
    pthread_mutex_unlock(&mutex);
}
