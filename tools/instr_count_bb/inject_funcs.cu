#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdint.h>
#include <stdio.h>

#include "utils/utils.h"

extern "C" __device__ __noinline__ void count_instrs(int num_instrs,
                                                     int count_warp_level,
                                                     uint64_t pcounter) {
    /* all the active threads will compute the active mask */
    const int active_mask = __ballot_sync(__activemask(), 1);

    /* each thread will get a lane id (get_lane_id is implemented in
     * utils/utils.h) */
    const int laneid = get_laneid();

    /* get the id of the first active thread */
    const int first_laneid = __ffs(active_mask) - 1;

    /* count all the active thread */
    const int num_threads = __popc(active_mask);

    /* only the first active thread will perform the atomic */
    if (first_laneid == laneid) {
        if (count_warp_level) {
            atomicAdd((unsigned long long*)pcounter, 1 * num_instrs);
        } else {
            atomicAdd((unsigned long long*)pcounter, num_threads * num_instrs);
        }
    }
}

extern "C" __device__ __noinline__ void count_pred_off(int predicate,
                                                       int count_warp_level,
                                                       uint64_t pcounter) {
    /* all the active threads will compute the active mask */
    const int active_mask = __ballot_sync(__activemask(), 1);

    /* each thread will get a lane id (get_lane_id is implemented in
     * utils/utils.h) */
    const int laneid = get_laneid();

    /* get the id of the first active thread */
    const int first_laneid = __ffs(active_mask) - 1;

    /* get predicate mask */
    const int predicate_mask = __ballot_sync(__activemask(), predicate);

    /* get mask of threads that have their predicate off */
    const int mask_off = active_mask ^ predicate_mask;

    /* count the number of threads that have their predicate off */
    const int num_threads_off = __popc(mask_off);

    /* only the first active thread updates the counter of predicated off
     * threads */
    if (first_laneid == laneid) {
        if (count_warp_level) {
            if (predicate_mask == 0) {
                atomicAdd((unsigned long long*)pcounter, 1);
            }
        } else {
            atomicAdd((unsigned long long*)pcounter, num_threads_off);
        }
    }
}
