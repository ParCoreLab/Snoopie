/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <pthread.h>
#include <stdint.h>
#include <stdio.h>
#include <map>
#include <unordered_set>

/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

/* provide some __device__ functions */
#include "utils/utils.h"

/* kernel id counter, maintained in system memory */
uint32_t kernel_id = 0;

/* total instruction counter, maintained in system memory, incremented by
 * "counter" every time a kernel completes  */
uint64_t tot_app_instrs = 0;

/* kernel instruction counter, updated by the GPU threads */
#define MAX_OPCODES (16 * 1024)
__managed__ uint64_t histogram[MAX_OPCODES];

/* global control variables for this tool */
uint32_t instr_begin_interval = 0;
uint32_t instr_end_interval = UINT32_MAX;
uint32_t ker_begin_interval = 0;
uint32_t ker_end_interval = UINT32_MAX;
int verbose = 0;
int count_warp_level = 1;
int exclude_pred_off = 0;

/* instruction to opcode map, used for final print of the opcodes */
std::map<std::string, int> instr_opcode_to_num_map;

/* a pthread mutex, used to prevent multiple kernels to run concurrently and
 * therefore to "corrupt" the counter variable */
pthread_mutex_t mutex;

/* nvbit_at_init() is executed as soon as the nvbit tool is loaded. We typically
 * do initializations in this call. In this case for instance we get some
 * environment variables values which we use as input arguments to the tool */
void nvbit_at_init() {
    /* just make sure all managed variables are allocated on GPU */
    setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);

    /* we get some environment variables that are going to be use to selectively
     * instrument (within a interval of kernel indexes and instructions). By
     * default we instrument everything. */
    GET_VAR_INT(
        instr_begin_interval, "INSTR_BEGIN", 0,
        "Beginning of the instruction interval where to apply instrumentation");
    GET_VAR_INT(
        instr_end_interval, "INSTR_END", UINT32_MAX,
        "End of the instruction interval where to apply instrumentation");
    GET_VAR_INT(ker_begin_interval, "KERNEL_BEGIN", 0,
                "Beginning of the kernel launch interval where to apply "
                "instrumentation");
    GET_VAR_INT(
        ker_end_interval, "KERNEL_END", UINT32_MAX,
        "End of the kernel launch interval where to apply instrumentation");
    GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool");
    GET_VAR_INT(count_warp_level, "COUNT_WARP_LEVEL", 1,
                "Count warp level or thread level instructions");
    GET_VAR_INT(exclude_pred_off, "EXCLUDE_PRED_OFF", 0,
                "Exclude predicated off instruction from count");

    std::string pad(100, '-');
    printf("%s\n", pad.c_str());
}

/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;

void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {
    /* Get related functions of the kernel (device function that can be
     * called by the kernel) */
    std::vector<hipFunction_t> related_functions =
        nvbit_get_related_functions(ctx, func);

    /* add kernel itself to the related function vector */
    related_functions.push_back(func);

    /* iterate on function */
    for (auto f : related_functions) {
        /* "recording" function was instrumented, if set insertion failed
         * we have already encountered this function */
        if (!already_instrumented.insert(f).second) {
            continue;
        }

        /* Get the vector of instruction composing the loaded CUFunction "func"
         */
        const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, func);

        /* Get kernel name */
        std::string kernel_name = nvbit_get_func_name(ctx, func);

        /* If verbose we print function name and number of" static" instructions
         */
        if (verbose) {
            printf("inspecting %s - num instrs %ld\n", kernel_name.c_str(),
                   instrs.size());
        }

        /* We iterate on the vector of instruction */
        for (auto i : instrs) {
            /* Check if the instruction falls in the interval where we want to
             * instrument */
            if (i->getIdx() < instr_begin_interval ||
                i->getIdx() >= instr_end_interval) {
                continue;
            }
            /* If verbose we print which instruction we are instrumenting */
            if (verbose) {
                i->print();
            }

            std::string opcode = i->getOpcode();
            if (instr_opcode_to_num_map.find(opcode) ==
                instr_opcode_to_num_map.end()) {
                size_t size = instr_opcode_to_num_map.size();
                instr_opcode_to_num_map[opcode] = size;
            }
            int instr_type = instr_opcode_to_num_map[opcode];

            /* Insert a call to "count_instrs" before the instruction "i" */
            nvbit_insert_call(i, "count_instrs", IPOINT_BEFORE);
            /* Add argument to the instrumentation function */
            if (exclude_pred_off) {
                /* pass predicate value */
                nvbit_add_call_arg_guard_pred_val(i);
            } else {
                /* pass always true */
                nvbit_add_call_arg_const_val32(i, 1);
            }

            /* add instruction type id */
            nvbit_add_call_arg_const_val32(i, instr_type);
            /* add count warps option */
            nvbit_add_call_arg_const_val32(i, count_warp_level);
            /* add pointer to counter location */
            nvbit_add_call_arg_const_val64(i, (uint64_t)histogram);
        }
    }
}

/* This call-back is triggered every time a CUDA event is encountered.
 * Here, we identify CUDA kernel launch events and reset the "counter" before
 * th kernel is launched, and print the counter after the kernel has completed
 * (we make sure it has completed by using hipDeviceSynchronize()). To
 * selectively run either the original or instrumented kernel we used
 * nvbit_enable_instrumented() before launching the kernel. */
void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char *name, void *params, hipError_t *pStatus) {
    /* Identify all the possible CUDA launch events */
    if (cbid == API_CUDA_cuLaunch || cbid == API_CUDA_cuLaunchKernel_ptsz ||
        cbid == API_CUDA_cuLaunchGrid || cbid == API_CUDA_cuLaunchGridAsync ||
        cbid == API_CUDA_cuLaunchKernel) {
        /* cast params to cuLaunch_params since if we are here we know these are
         * the right parameters type */
        cuLaunch_params *p = (cuLaunch_params *)params;

        if (!is_exit) {
            /* if we are entering in a kernel launch:
             * 1. Lock the mutex to prevent multiple kernels to run concurrently
             * (overriding the counter) in case the user application does that
             * 2. Instrument the function if needed
             * 3. Select if we want to run the instrumented or original
             * version of the kernel
             * 4. Reset the kernel instruction counter */

            pthread_mutex_lock(&mutex);
            instrument_function_if_needed(ctx, p->f);

            if (kernel_id >= ker_begin_interval &&
                kernel_id < ker_end_interval) {
                nvbit_enable_instrumented(ctx, p->f, true);
            } else {
                nvbit_enable_instrumented(ctx, p->f, false);
            }
            memset(histogram, 0, sizeof(uint64_t) * MAX_OPCODES);
        } else {
            /* if we are exiting a kernel launch:
             * 1. Wait until the kernel is completed using
             * hipDeviceSynchronize()
             * 2. Get number of thread blocks in the kernel
             * 3. Print the thread instruction counters
             * 4. Release the lock*/
            CUDA_SAFECALL(hipDeviceSynchronize());
            int num_ctas = 0;
            if (cbid == API_CUDA_cuLaunchKernel_ptsz ||
                cbid == API_CUDA_cuLaunchKernel) {
                cuLaunchKernel_params *p2 = (cuLaunchKernel_params *)params;
                num_ctas = p2->gridDimX * p2->gridDimY * p2->gridDimZ;
            }
            uint64_t counter = 0;
            for (auto a : instr_opcode_to_num_map) {
                if (histogram[a.second] != 0) {
                    counter += histogram[a.second];
                }
            }
            tot_app_instrs += counter;
            printf(
                "kernel %d - %s - #thread-blocks %d,  kernel "
                "instructions %ld, total instructions %ld\n",
                kernel_id++, nvbit_get_func_name(ctx, p->f), num_ctas, counter,
                tot_app_instrs);

            for (auto a : instr_opcode_to_num_map) {
                if (histogram[a.second] != 0) {
                    printf("  %s = %ld\n", a.first.c_str(),
                           histogram[a.second]);
                }
            }
            pthread_mutex_unlock(&mutex);
        }
    }
}
